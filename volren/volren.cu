#include "hip/hip_runtime.h"
#include "volren.cuh"
#include "bvh.cuh"
#include "common.cuh"
  
#if WITH_CUDA
texture<float4, 1, hipReadModeElementType> texTransferFunc;
#endif


__device__ __host__
bool QuadNodeD_insideQuad(const QuadNodeD &q, float x, float y)
{
  return x >= q.Ax && x < q.Bx && y >= q.Ay && y < q.By;
}

__device__ __host__
bool QuadNodeD_insideTriangle(const QuadNodeD &q, float x, float y, float3 &lambda)
{
  lambda.x = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) /
          ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  lambda.y = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) /
         ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  lambda.z = 1.0 - lambda.x - lambda.y;
  // fprintf(stderr, "barycentric=%f, %f, %f\n", lambda.x, lambda.y, lambda.z);
  return lambda.x >= 0 && lambda.y >= 0 && lambda.z >= 0;
}

__device__ __host__
int QuadNodeD_locatePoint_recursive(const QuadNodeD *q, const QuadNodeD *nodes, float x, float y, float3 &lambda)
{
  if (q->triangleId >= 0) { //leaf node
    bool succ = QuadNodeD_insideTriangle(*q, x, y, lambda);
    if (succ) return q->triangleId;
  } else if (QuadNodeD_insideQuad(*q, x, y)) {
    for (int j=0; j<4; j++) {
      if (q->childrenIds[j] > 0) {
        int result = QuadNodeD_locatePoint_recursive(&nodes[q->childrenIds[j]], nodes, x, y, lambda);
        if (result >= 0) return result;
      }
    }
  }
  return -1;
}

__device__ __host__
int QuadNodeD_locatePoint(QuadNodeD *nodes, float x, float y, float3 &lambda)
{
  // float lambda.x, lambda.y, lambda.z;
  static const int maxStackSize = 64;
  int stack[maxStackSize];
  int stackPos = 0;
  stack[stackPos++] = 0; // push root

  while (stackPos > 0) {
    const int i = stack[--stackPos]; // pop
    const QuadNodeD &q = nodes[i];

    // fprintf(stderr, "D_checking node %d, %f, %f, %f, %f\n", i, q.Ax, q.Ay, q.Bx, q.By);
    // fprintf(stderr, "D_checking node %d\n", i);

    if (q.triangleId >= 0) { // leaf node
      bool succ = QuadNodeD_insideTriangle(q, x, y, lambda);
      if (succ) return q.triangleId;
    } else if (QuadNodeD_insideQuad(q, x, y)) { // non-leaf node
      for (int j=0; j<4; j++) {
        if (q.childrenIds[j] > 0)
          stack[stackPos++] = q.childrenIds[j];
      }
    }
  }
  return -1;
}

__device__ __host__
int QuadNodeD_locatePoint_coherent(QuadNodeD *bvh, int last_nid, float x, float y, float3 &lambda)
{
  // check if last_nid is valid
  if (last_nid<0) return QuadNodeD_locatePoint(bvh, x, y, lambda);

  // check if in the same triangle
  if (QuadNodeD_insideTriangle(bvh[last_nid], x, y, lambda)) return last_nid;

  // TODO: check if in the neighbors of last_nid

  return QuadNodeD_locatePoint(bvh, x, y, lambda);
}

__device__ __host__
float QuadNodeD_sample(QuadNodeD* bvh, int nid, float3 lambda, float *data) {
  const QuadNodeD &q = bvh[nid];

  return lambda.x * data[q.i0] 
    + lambda.y * data[q.i1]
    + lambda.z * data[q.i2];
}

__device__ __host__
float interpolateXGC(QuadNodeD *bvh, float3 pos, float *data)
{
  // compute cylindrical coordiates
  float r = sqrt(pos.x*pos.x + pos.y + pos.y);
  float theta = atan2(pos.y, pos.x);
  // float z = pos.z;
  
  return 0;
}

__device__ __host__ 
static float4 value2color(float value, float *tf, float2 trans)
{
  static const int n = 256;
  static const float delta = 1.f / (n-1);
  const float x = clamp(value * trans.x + trans.y, 0.f, 0.9999f);
  const int i = max((int)(x*delta), n-2) , j = i + 1;
  const float beta = x - i*delta, alpha = 1 - beta;

  return make_float4(
      alpha * tf[i*4] + beta * tf[j*4], 
      alpha * tf[i*4+1] + beta * tf[j*4+1], 
      alpha * tf[i*4+2] + beta * tf[j*4+2], 
      alpha * tf[i*4+3] + beta * tf[j*4+3]);
}

  
template <int SHADING>
__device__ __host__ static void rc(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int nNodes,               // number of nodes 
        float *data,              // volume data in unstructured mesh
        QuadNodeD *bvh,
        float *disp,
        float *tf,
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize, 
        float tnear, float tfar)
{
  const float pi = 3.141592654f;
  const float pi2 = 2*pi;
  float4 src;
  // float3 N, L = make_float3(1, 0, 0), V = rayD; 
  // float3 Ka = make_float3(0.04), 
  //        Kd = make_float3(0.3), 
  //        Ks = make_float3(0.2); 
  // const float delta = 0.5f / dsz.x;   // for shading 
  float3 pos;
  float t = tnear;

  int last_nid = -1, nid;

  while (t < tfar) {
    pos = rayO + rayD*t;

    // cylindar coordinates
    float r = sqrt(pos.x*pos.x + pos.y*pos.y);
    float phi = atan2(pos.y, pos.x) + pi;
    float z = pos.z;

    float3 lambda;
    nid = QuadNodeD_locatePoint(bvh, r, z, lambda);
    // nid = QuadNodeD_locatePoint_coherent(bvh, last_nid, r, z, lambda);

    if (nid != -1) {
      const float unitAngle = pi2/nPhi;

      int p0 = (int)(phi/unitAngle)%nPhi;
      int p1 = (p0+1)%nPhi;
    
      float alpha = (phi - unitAngle*p0) / unitAngle;
      float v0 = QuadNodeD_sample(bvh, nid, lambda, data + nNodes*p0); //  + nNodes*p0);
      float v1 = QuadNodeD_sample(bvh, nid, lambda, data + nNodes*p1); //  + nNodes*p1);

      float value = (1-alpha)*v0 + alpha*v1;
      
      // sample = interpolateXGC(bvh, pos, data); 
      // sample = QuadNodeD_sample(bvh, x, y, data);
      // sample = tex3Dtrans<DataType, readMode, TRANSFORM>(texVolume, trans, coords); 
      // src = make_float4(sample, 1.0-sample, 0.0, 0.9);
      // sample = pow(1.f - sample, 2.f); 
      // src = make_float4(sample*2, 1.f-sample*2, 0.0, sample*0.4); 
      // src = make_float4(lambda.x, lambda.y, lambda.z, 0.5);
      
      // src = tex1D(texTransferFunc, value * trans.x + trans.y);
      src = value2color(value, tf, trans);

      // float v = clamp(value*0.01, -0.5f, 0.5f);
      // src = make_float4(v+0.5, 0.5-v, 0, min(1.f, v*v*10));

#if 0
      if (SHADING) {
        float3 lit; 
        N = gradient(texVolume, coords, delta); 
        lit = cook(N, V, L, Ka, Kd, Ks); 
        src.x += lit.x; 
        src.y += lit.y; 
        src.z += lit.z; 
      }
#endif
      
      src.w = 1.f - pow(1.f - src.w, stepsize); // alpha correction  

      dst.x += (1.0 - dst.w) * src.x * src.w;
      dst.y += (1.0 - dst.w) * src.y * src.w;
      dst.z += (1.0 - dst.w) * src.z * src.w;
      dst.w += (1.0 - dst.w) * src.w;
    }
    
    if (dst.w > 0.98) return; // early ray termination
    t += stepsize; 
  }
  // dst.x = 1; dst.y = 0; dst.z = 0; dst.w = 1;
}

template <int SHADING>
__device__ __host__ static void raycasting(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int nNodes,               // number of nodes 
        float *data,              // volume data in unstructured mesh
        QuadNodeD *bvh,
        float *disp,
        float *tf, 
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize)           // stepsize
{
  const QuadNodeD &root = bvh[0];
  const float innerRadius = root.Ax, outerRadius = root.Bx;
  // const float innerRadius = 1.0, outerRadius = 1.2;
  const float z0 = root.Ay, z1 = root.By; 
  float tnear0, tfar0, tnear1, tfar1;
  bool b0 = intersectCylinder(rayO, rayD, tnear0, tfar0, outerRadius, z0, z1), 
       b1 = intersectCylinder(rayO, rayD, tnear1, tfar1, innerRadius, z0, z1);
  
#if 1
  if (b0 && (!b1))
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, disp, tf, trans, rayO, rayD, stepsize, tnear0, tfar0);
  else if (b0 && b1) {
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, disp, tf, trans, rayO, rayD, stepsize, tnear0, tnear1);
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, disp, tf, trans, rayO, rayD, stepsize, tfar1, tfar0);
  }
#else
  if (b1)
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, trans, rayO, rayD, stepsize, tnear1, tfar1);
#endif
}

#if WITH_CUDA
__global__ static void test_point_locator_kernel(
    int *output, 
    float x, float y,
    QuadNodeD *bvh)
{
  float3 lambda;
  *output = QuadNodeD_locatePoint(bvh, x, y, lambda);
}
#endif

__device__ __host__ bool setup_ray(
    int *viewport, 
    float *invmvp, 
    uint x, uint y,
    float3 &rayO, float3 &rayD)
{
  float coord[4], obj0[4], obj1[4]; 
  coord[0] = (x-viewport[0])*2.f / viewport[2] - 1.f; 
  coord[1] = (y-viewport[1])*2.f / viewport[3] - 1.f; 
  coord[2] = -1.0; 
  coord[3] = 1.0;

  mulmatvec(invmvp, coord, obj0); 
  coord[2] = 1.0; 
  mulmatvec(invmvp, coord, obj1); 
  if (obj0[3] == 0.f || obj1[3] == 0.f) return false; 

  for (int i=0; i<3; i++)
      obj0[i] /= obj0[3], obj1[i] /= obj1[3]; 

  rayO = make_float3(obj0[0], obj0[1], obj0[2]);
  rayD = normalize(make_float3(obj1[0]-obj0[0], obj1[1]-obj0[1], obj1[2]-obj0[2]));
  return true;
}

#if WITH_CUDA
template <int SHADING>
__global__ static void raycasting_kernel(
        unsigned char *output_rgba8,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int nNode, 
        float *data, 
        QuadNodeD *bvh,
        float *disp,
        float *tf,
        float2 trans, 
        float stepsize)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= viewport[2] || y>= viewport[3]) return;
 
  float3 rayO, rayD;
  setup_ray(viewport, invmvp, x, y, rayO, rayD);

  float4 dst = make_float4(0.f); 
  raycasting<SHADING>(dst, nPhi, nNode, data, bvh, disp, tf, trans, rayO, rayD, stepsize);

  output_rgba8[(y*viewport[2]+x)*4+0] = dst.x * 255;
  output_rgba8[(y*viewport[2]+x)*4+1] = dst.y * 255;
  output_rgba8[(y*viewport[2]+x)*4+2] = dst.z * 255;
  output_rgba8[(y*viewport[2]+x)*4+3] = dst.w * 255;

#if 0
  // GL_ONE_MINUS_DST_ALPHA, GL_ONE
  float w0 = 1-output[(y*viewport[2]+x)*4+3]; //, w1 = 1; make the compiler happy :)
  output[(y*viewport[2]+x)*4+0] += w0* dst.x;
  output[(y*viewport[2]+x)*4+1] += w0* dst.y;
  output[(y*viewport[2]+x)*4+2] += w0* dst.z;
  output[(y*viewport[2]+x)*4+3] += w0* dst.w;
#endif
}
#endif

template <int SHADING>
static void raycasting_cpu(
        unsigned char *output_rgba8,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int nNode, 
        float *data, 
        QuadNodeD *bvh,
        float *disp,
        float *tf,
        float2 trans, 
        float stepsize)
{
#pragma omp parallel for collapse(2)
  for (uint x = 0; x < viewport[2]; x ++) {
    for (uint y = 0; y < viewport[3]; y ++) {
      float3 rayO, rayD;
      setup_ray(viewport, invmvp, x, y, rayO, rayD);

      float4 dst = make_float4(0.f); 
      raycasting<SHADING>(dst, nPhi, nNode, data, bvh, disp, tf, trans, rayO, rayD, stepsize);

      output_rgba8[(y*viewport[2]+x)*4+0] = dst.x * 255;
      output_rgba8[(y*viewport[2]+x)*4+1] = dst.y * 255;
      output_rgba8[(y*viewport[2]+x)*4+2] = dst.z * 255;
      output_rgba8[(y*viewport[2]+x)*4+3] = dst.w * 255;
    }
  }
}

/////////////////////////////
extern "C" {

void rc_test_point_locator(ctx_rc *ctx, float x, float y)
{
#if WITH_CUDA
  test_point_locator_kernel<<<1, 1>>>(
      (int*)ctx->d_output_rgba8, 
      x, y, ctx->d_bvh);

  int nid;
  hipMemcpy(&nid, ctx->d_output_rgba8, sizeof(int), hipMemcpyDeviceToHost);
  fprintf(stderr, "[rc_test_point_locator] x={%f, %f}, nid=%d\n", x, y, nid);
#endif
}

void rc_render(ctx_rc *ctx)
{
#if WITH_CUDA
  const dim3 blockSize(16, 16); 
  const dim3 gridSize = dim3(iDivUp(ctx->viewport[2], blockSize.x), iDivUp(ctx->viewport[3], blockSize.y));

  hipMemcpy(ctx->d_viewport, ctx->viewport, sizeof(int)*4, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_viewport), ctx->viewport, sizeof(int)*4);
  hipMemcpy(ctx->d_invmvp, ctx->invmvp, sizeof(float)*16, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_invmvp), ctx->invmvp, sizeof(float)*16);
 
  raycasting_kernel<0><<<gridSize, blockSize>>>(
          ctx->d_output_rgba8,
          ctx->d_viewport, 
          ctx->d_invmvp,
          ctx->nPhi, 
          ctx->nNodes,
          ctx->d_data, 
          ctx->d_bvh,
          ctx->d_disp,
          ctx->d_tf,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);

  hipDeviceSynchronize();
  checkLastCudaError("[rc_render]");
#endif
}

void rc_render_cpu(ctx_rc *ctx)
{
  raycasting_cpu<0>(
          (unsigned char*)ctx->h_output,
          ctx->viewport, 
          ctx->invmvp,
          ctx->nPhi, 
          ctx->nNodes,
          ctx->h_data, 
          ctx->h_bvh,
          ctx->h_disp,
          ctx->h_tf,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);
}

void rc_bind_bvh(ctx_rc *ctx, int nQuadNodes, QuadNodeD *bvh)
{
#if WITH_CUDA
  ctx->h_bvh = bvh;

  if (ctx->d_bvh != NULL)
    hipFree(ctx->d_bvh);

  hipMalloc((void**)&ctx->d_bvh, sizeof(QuadNodeD)*nQuadNodes);
  hipMemcpy(ctx->d_bvh, bvh, sizeof(QuadNodeD)*nQuadNodes, hipMemcpyHostToDevice);
#endif
}

void rc_set_default_tf(ctx_rc *ctx)
{
  float *tf = ctx->h_tf;
  for (int i=0; i<size_tf; i++) {
    float x = (float)i / size_tf;
    tf[i*4] = x;
    tf[i*4+1] = 1-x;
    tf[i*4+2] = 0.f;
    tf[i*4+3] = 0.2;
  }
  rc_set_tf(ctx, tf);
}

void rc_set_tf(ctx_rc *ctx, float *tf) 
{
  if (tf != ctx->h_tf)
    memcpy(ctx->h_tf, tf, sizeof(float)*size_tf*4);

#if WITH_CUDA
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

  hipMemcpy(ctx->d_tf, tf, sizeof(float)*size_tf*4, hipMemcpyHostToDevice);
  hipMemcpyToArray( ctx->d_tfArray, 0, 0, tf, size_tf*4, hipMemcpyHostToDevice ); 

  texTransferFunc.normalized = true; 
  texTransferFunc.filterMode = hipFilterModeLinear; 
  texTransferFunc.addressMode[0] = hipAddressModeClamp; 
  hipBindTextureToArray(texTransferFunc, ctx->d_tfArray, channelDesc); 

  checkLastCudaError("[rc_set_tf]");
#endif
}

void rc_bind_disp(ctx_rc *ctx, int nNodes, float *disp)
{
  ctx->h_disp = disp;
#if WITH_CUDA
  if (ctx->d_disp == NULL)
    hipMalloc((void**)&ctx->d_disp, sizeof(float)*nNodes*2);
  hipMemcpy(ctx->d_disp, disp, sizeof(float)*nNodes*2, hipMemcpyHostToDevice);
#endif
}

void rc_bind_data(ctx_rc *ctx, int nNodes, int nPhi, float *data)
{
  ctx->h_data = data;
  ctx->nNodes = nNodes;
  ctx->nPhi = nPhi;
#if WITH_CUDA
  if (ctx->d_data == NULL)
    hipMalloc((void**)&ctx->d_data, sizeof(float)*nNodes*nPhi);
  hipMemcpy(ctx->d_data, data, sizeof(float)*nNodes*nPhi, hipMemcpyHostToDevice);
#endif
}

void rc_create_ctx(ctx_rc **ctx)
{
  *ctx = (ctx_rc*)malloc(sizeof(ctx_rc));
  memset(*ctx, 0, sizeof(ctx_rc));

  (*ctx)->h_tf = (float*)malloc(sizeof(float)*size_tf*4);

  const size_t max_npx = 4096*4096;

#if WITH_CUDA
  hipSetDevice(0);
  hipMalloc((void**)&((*ctx)->d_output_rgba8), 4*max_npx); 
  (*ctx)->h_output = malloc(4*max_npx);
  
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  hipMallocArray( &(*ctx)->d_tfArray, &channelDesc, size_tf*4, 1 ); 

  hipMalloc((void**)&((*ctx)->d_tf), sizeof(float)*size_tf*4);
  hipMalloc((void**)&((*ctx)->d_viewport), sizeof(int)*4);
  hipMalloc((void**)&((*ctx)->d_invmvp), sizeof(float)*16);
  
  checkLastCudaError("[rc_init]");
#endif
}

void rc_destroy_ctx(ctx_rc **ctx)
{
  // TODO: free any resources
#if WITH_CUDA
  hipFree((*ctx)->d_output_rgba8);
#endif
  free((*ctx)->h_output);
  free((*ctx)->h_tf);
  free(*ctx); 
  *ctx = NULL; 
}

void rc_set_viewport(ctx_rc *ctx, int x, int y, int w, int h)
{
  ctx->viewport[0] = x; 
  ctx->viewport[1] = y; 
  ctx->viewport[2] = w; 
  ctx->viewport[3] = h; 
}

void rc_set_range(ctx_rc *ctx, float a, float b)
{
  float c = 1.f/(b-a);
  ctx->trans[0] = c; 
  ctx->trans[1] = -a*c; 
}

void rc_set_stepsize(ctx_rc *ctx, float stepsize)
{
  ctx->stepsize = stepsize;
}

void rc_set_invmvpf(ctx_rc *ctx, float *invmvp)
{
  memcpy(ctx->invmvp, invmvp, sizeof(float)*16); 
}

void rc_set_invmvpd(ctx_rc *ctx, double *invmvp)
{
  for (int i=0; i<16; i++) {
    ctx->invmvp[i] = invmvp[i]; 
  }
}

void rc_clear_output(ctx_rc *ctx)
{
#if WITH_CUDA
  hipMemset(ctx->d_output_rgba8, 0, 4*sizeof(float)*ctx->viewport[2]*ctx->viewport[3]);
#endif
}

void rc_copy_output_to_host(ctx_rc *ctx)
{
#if WITH_CUDA
  hipMemcpy(ctx->h_output, ctx->d_output_rgba8, 4*sizeof(float)*ctx->viewport[2]*ctx->viewport[3], hipMemcpyDeviceToHost); 
#endif
}

} // extern "C" 
/////////////////

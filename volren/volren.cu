#include "hip/hip_runtime.h"
#include "volren.cuh"
#include "bvh.cuh"
#include "common.cuh"
  
#if WITH_CUDA
texture<float4, 1, hipReadModeElementType> texTransferFunc;
#endif


__device__ __host__
inline bool QuadNodeD_insideQuad(const QuadNodeD &q, float x, float y)
{
  return x >= q.Ax && x < q.Bx && y >= q.Ay && y < q.By;
}

__device__ __host__
inline bool QuadNodeD_insideTriangle(const QuadNodeD &q, float x, float y, float3 &lambda, float *invdet)
{
#if 0
  lambda.x = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) /
          ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  lambda.y = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) /
         ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
#endif
  const float d = invdet[q.triangleId];
  lambda.x = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) * d; 
  lambda.y = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) * d;
  lambda.z = 1.0 - lambda.x - lambda.y;
  // fprintf(stderr, "barycentric=%f, %f, %f\n", lambda.x, lambda.y, lambda.z);
  return lambda.x >= 0 && lambda.y >= 0 && lambda.z >= 0;
}

__device__ __host__
inline int QuadNodeD_locatePoint_recursive(const QuadNodeD *q, const QuadNodeD *nodes, float x, float y, float3 &lambda, float *invdet)
{
  if (q->triangleId >= 0) { //leaf node
    bool succ = QuadNodeD_insideTriangle(*q, x, y, lambda, invdet);
    if (succ) return q->triangleId;
  } else if (QuadNodeD_insideQuad(*q, x, y)) {
    for (int j=0; j<4; j++) {
      if (q->childrenIds[j] > 0) {
        int result = QuadNodeD_locatePoint_recursive(&nodes[q->childrenIds[j]], nodes, x, y, lambda, invdet);
        if (result >= 0) return result;
      }
    }
  }
  return -1;
}

__device__ __host__
inline int QuadNodeD_locatePoint(QuadNodeD *nodes, float x, float y, float3 &lambda, float *invdet)
{
  // float lambda.x, lambda.y, lambda.z;
  static const int maxStackSize = 64;
  int stack[maxStackSize];
  int stackPos = 0;
  stack[stackPos++] = 0; // push root

  while (stackPos > 0) {
    const int i = stack[--stackPos]; // pop
    const QuadNodeD &q = nodes[i];

    // fprintf(stderr, "D_checking node %d, %f, %f, %f, %f\n", i, q.Ax, q.Ay, q.Bx, q.By);
    // fprintf(stderr, "D_checking node %d\n", i);

    if (q.triangleId >= 0) { // leaf node
      bool succ = QuadNodeD_insideTriangle(q, x, y, lambda, invdet);
      if (succ) return q.triangleId;
    } else if (QuadNodeD_insideQuad(q, x, y)) { // non-leaf node
      for (int j=0; j<4; j++) {
        if (q.childrenIds[j] > 0)
          stack[stackPos++] = q.childrenIds[j];
      }
    }
  }
  return -1;
}

__device__ __host__
inline int QuadNodeD_locatePoint_coherent(QuadNodeD *bvh, int last_nid, float x, float y, float3 &lambda, float *invdet)
{
  // check if last_nid is valid
  if (last_nid<0) return QuadNodeD_locatePoint(bvh, x, y, lambda, invdet);

  // check if in the same triangle
  if (QuadNodeD_insideTriangle(bvh[last_nid], x, y, lambda, invdet)) return last_nid;

  // TODO: check if in the neighbors of last_nid

  return QuadNodeD_locatePoint(bvh, x, y, lambda, invdet);
}

__device__ __host__
inline float QuadNodeD_sample(int i0, int i1, int i2, float3 lambda, float *data) {
  return lambda.x * data[i0] + lambda.y * data[i1] + lambda.z * data[i2];
}

__device__ __host__
inline float2 QuadNodeD_sample2(int i0, int i1, int i2, float3 lambda, float *data) {
  return make_float2(lambda.x * data[i0*2] + lambda.y * data[i1*2] + lambda.z * data[i2*2],
      lambda.x * data[i0*2+1] + lambda.y * data[i1*2+1] + lambda.z * data[i2*2+1]);
}

__device__ __host__
inline float QuadNodeD_sample(QuadNodeD* bvh, int nid, float3 lambda, float *data) {
  const QuadNodeD &q = bvh[nid];
  return lambda.x * data[q.i0] + lambda.y * data[q.i1] + lambda.z * data[q.i2];
}

__device__ __host__
inline float2 QuadNodeD_sample2(QuadNodeD* bvh, int nid, float3 lambda, float *data) {
  const QuadNodeD &q = bvh[nid];
  return QuadNodeD_sample2(q.i0, q.i1, q.i2, lambda, data);
  // return make_float2(lambda.x * data[q.i0*2] + lambda.y * data[q.i1*2] + lambda.z * data[q.i2*2],
  //     lambda.x * data[q.i0*2+1] + lambda.y * data[q.i1*2+1] + lambda.z * data[q.i2*2+1]);
}

__device__ __host__
inline int interpolateXGC(float &value, QuadNodeD *bvh, float3 p, int nPhi, int nNodes, float *data, float *invdet)
{
  static const float pi = 3.141592654f;
  static const float pi2 = 2*pi;
  
  // cylindar coordinates
  float r = sqrt(p.x*p.x + p.y*p.y);
  float phi = atan2(p.y, p.x) + pi;
  float z = p.z;
  float3 lambda;
  
  int nid = QuadNodeD_locatePoint(bvh, r, z, lambda, invdet);
  if (nid == -1) return nid; 
  
  const QuadNodeD &q = bvh[nid];

  const float deltaAngle = pi2/nPhi;
  int p0 = (int)(phi/deltaAngle)%nPhi;
  int p1 = (p0+1)%nPhi;

  float alpha = (phi - deltaAngle*p0) / deltaAngle;
  // float v0 = QuadNodeD_sample(bvh, nid, lambda, data + nNodes*p0);
  // float v1 = QuadNodeD_sample(bvh, nid, lambda, data + nNodes*p1);
  float v0 = QuadNodeD_sample(q.i0, q.i1, q.i2, lambda, data + nNodes*p0); //  + nNodes*p0);
  float v1 = QuadNodeD_sample(q.i0, q.i1, q.i2, lambda, data + nNodes*p1); //  + nNodes*p1);
  // float dr0 = QuadNodeD

  value = (1-alpha)*v0 + alpha*v1;
  return nid;
}

__device__ __host__
inline int interpolateXGC2(float &value, QuadNodeD *bvh, float3 p, int nPhi, int nNodes, float *data, float *disp, float *invdet)
{
  static const float pi = 3.141592654f;
  static const float pi2 = 2*pi;
  
  // cylindar coordinates
  float r = sqrt(p.x*p.x + p.y*p.y);
  float phi = atan2(p.y, p.x) + pi;
  float z = p.z;
  float3 lambda;
  
  int nid = QuadNodeD_locatePoint(bvh, r, z, lambda, invdet);
  if (nid == -1) return nid; 
      
  const float deltaAngle = pi2/nPhi;

  int p0 = (int)(phi/deltaAngle)%nPhi;
  int p1 = (p0+1)%nPhi;
  float alpha = (phi - deltaAngle*p0) / deltaAngle;

  // interpolate disp
  const QuadNodeD &q = bvh[nid];
  float dx = lambda.x * disp[q.i0*2] + lambda.y * disp[q.i1*2] + lambda.z * disp[q.i2*2];
  float dy = lambda.x * disp[q.i0*2+1] + lambda.y * disp[q.i1*2+1] + lambda.z * disp[q.i2*2+1];
 
  float3 lambda0, lambda1;
  int nid0 = QuadNodeD_locatePoint_coherent(bvh, nid, r+dx*(1-alpha), z+dy*(1-alpha), lambda0, invdet);
  int nid1 = QuadNodeD_locatePoint_coherent(bvh, nid, r+dx*alpha, z+dy*alpha, lambda1, invdet);
  if (nid0 == -1 || nid1 == -1) {
    // fprintf(stderr, "nid=%d, nid0=%d, nid1=%d, dx=%f, dy=%f\n", nid, nid0, nid1, dx, dy);
    return -1;
  }

  float v0 = QuadNodeD_sample(bvh, nid0, lambda0, data + nNodes*p0); //  + nNodes*p0);
  float v1 = QuadNodeD_sample(bvh, nid1, lambda1, data + nNodes*p1); //  + nNodes*p1);

  // if (alpha<0 || alpha>=1) fprintf(stderr, "%f\n", alpha);
  
  value = (1-alpha)*v0 + alpha*v1;
  return nid;
}

__device__ __host__ 
static inline float4 value2color(float value, float *tf, float2 trans)
{
  const float x = clamp(value * trans.x + trans.y, 0.f, 1.f);
#if 0
  float v = x-0.5;
  return make_float4(x, 1-x, 0, fminf(0.999f, v*v*40));
#else 
  static const int n = 256;
  static const float delta = 1.f / (n-1);
  const int i = min((int)(x*(n-1)), n-2) , j = i + 1;
  const float beta = x - i*delta, alpha = 1 - beta;

  return make_float4(
      alpha * tf[i*4] + beta * tf[j*4], 
      alpha * tf[i*4+1] + beta * tf[j*4+1], 
      alpha * tf[i*4+2] + beta * tf[j*4+2], 
      alpha * tf[i*4+3] + beta * tf[j*4+3]);
#endif
}

  
template <int SHADING>
__device__ __host__ static inline void rc(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int nNodes,               // number of nodes 
        float *data,              // volume data in unstructured mesh
        float *grad,              // gradient
        QuadNodeD *bvh,
        float *disp,
        float *invdet,
        float *tf,
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize, 
        float tnear, float tfar)
{
  float4 src;
  // float3 N, L = make_float3(1, 0, 0), V = rayD; 
  // float3 Ka = make_float3(0.04), 
  //        Kd = make_float3(0.3), 
  //        Ks = make_float3(0.2); 
  // const float delta = 0.5f / dsz.x;   // for shading 
  float3 pos;
  float value;
  float t = tnear;

  while (t < tfar) {
    pos = rayO + rayD*t;

    const int nid = interpolateXGC(value, bvh, pos, nPhi, nNodes, data, invdet);
    // const int nid = interpolateXGC2(value, bvh, pos, nPhi, nNodes, data, disp, invdet);
    if (nid >= 0) {
      src = value2color(value, tf, trans);

#if 0
      if (SHADING) {
        float3 lit; 
        N = gradient(texVolume, coords, delta); 
        lit = cook(N, V, L, Ka, Kd, Ks); 
        src.x += lit.x; 
        src.y += lit.y; 
        src.z += lit.z; 
      }
#endif
      
      src.w = 1.f - pow(1.f - src.w, stepsize*4); // alpha correction  

      dst.x += (1.0 - dst.w) * src.x * src.w;
      dst.y += (1.0 - dst.w) * src.y * src.w;
      dst.z += (1.0 - dst.w) * src.z * src.w;
      dst.w += (1.0 - dst.w) * src.w;
    }
    
    // if (dst.w > 0.98) return; // early ray termination
    t += stepsize; 
  }
  // dst.x = 1; dst.y = 0; dst.z = 0; dst.w = 1;
}

template <int SHADING>
__device__ __host__ static inline void raycasting(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int nNodes,               // number of nodes 
        float *data,              // volume data in unstructured mesh
        float *grad,
        QuadNodeD *bvh,
        float *disp,
        float *invdet,
        float *tf, 
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize)           // stepsize
{
  const QuadNodeD &root = bvh[0];
  const float innerRadius = root.Ax, outerRadius = root.Bx;
  const float z0 = root.Ay, z1 = root.By; 
  float tnear0=-FLT_MAX, tfar0=FLT_MAX, tnear1=-FLT_MAX, tfar1=FLT_MAX;
  bool b0 = intersectCylinder(rayO, rayD, tnear0, tfar0, outerRadius, z0, z1), 
       b1 = intersectCylinder(rayO, rayD, tnear1, tfar1, innerRadius, z0, z1);
  
#if 1
  if (b0 && (!b1))
    rc<SHADING>(dst, nPhi, nNodes, data, grad, bvh, disp, invdet, tf, trans, rayO, rayD, stepsize, tnear0, tfar0);
  else if (b0 && b1) {
    rc<SHADING>(dst, nPhi, nNodes, data, grad, bvh, disp, invdet, tf, trans, rayO, rayD, stepsize, tnear0, tnear1);
    rc<SHADING>(dst, nPhi, nNodes, data, grad, bvh, disp, invdet, tf, trans, rayO, rayD, stepsize, tfar1, tfar0);
  }
#else
  if (b0) {
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, disp, tf, trans, rayO, rayD, stepsize, tnear0, tfar0);
  }
#endif
}

#if WITH_CUDA
__global__ static void test_point_locator_kernel(
    int *output, 
    float x, float y,
    QuadNodeD *bvh,
    float *invdet)
{
  float3 lambda;
  *output = QuadNodeD_locatePoint(bvh, x, y, lambda, invdet);
}
#endif

__device__ __host__ inline bool setup_ray(
    int *viewport, 
    float *invmvp, 
    uint x, uint y,
    float3 &rayO, float3 &rayD)
{
  float coord[4], obj0[4], obj1[4]; 
  coord[0] = (x-viewport[0])*2.f / viewport[2] - 1.f; 
  coord[1] = (y-viewport[1])*2.f / viewport[3] - 1.f; 
  coord[2] = -1.0; 
  coord[3] = 1.0;

  mulmatvec(invmvp, coord, obj0); 
  coord[2] = 1.0; 
  mulmatvec(invmvp, coord, obj1); 
  if (obj0[3] == 0.f || obj1[3] == 0.f) return false; 

  for (int i=0; i<3; i++)
      obj0[i] /= obj0[3], obj1[i] /= obj1[3]; 

  rayO = make_float3(obj0[0], obj0[1], obj0[2]);
  rayD = normalize(make_float3(obj1[0]-obj0[0], obj1[1]-obj0[1], obj1[2]-obj0[2]));
  return true;
}

#if WITH_CUDA
template <int SHADING>
__global__ static void raycasting_kernel(
        unsigned char *output_rgba8,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int nNodes, 
        float *data, 
        float *grad,
        QuadNodeD *bvh,
        float *disp,
        float *invdet,
        float *tf,
        float2 trans, 
        float stepsize)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= viewport[2] || y>= viewport[3]) return;
 
  float3 rayO, rayD;
  setup_ray(viewport, invmvp, x, y, rayO, rayD);

  float4 dst = make_float4(0.f); 
  raycasting<SHADING>(dst, nPhi, nNodes, data, grad, bvh, disp, invdet, tf, trans, rayO, rayD, stepsize);

  output_rgba8[(y*viewport[2]+x)*4+0] = dst.x * 255;
  output_rgba8[(y*viewport[2]+x)*4+1] = dst.y * 255;
  output_rgba8[(y*viewport[2]+x)*4+2] = dst.z * 255;
  output_rgba8[(y*viewport[2]+x)*4+3] = dst.w * 255;

  // if (y == 300) printf("%f\n", dst.w);

#if 0
  // GL_ONE_MINUS_DST_ALPHA, GL_ONE
  float w0 = 1-output[(y*viewport[2]+x)*4+3]; //, w1 = 1; make the compiler happy :)
  output[(y*viewport[2]+x)*4+0] += w0* dst.x;
  output[(y*viewport[2]+x)*4+1] += w0* dst.y;
  output[(y*viewport[2]+x)*4+2] += w0* dst.z;
  output[(y*viewport[2]+x)*4+3] += w0* dst.w;
#endif
}
#endif

template <int SHADING>
static void raycasting_cpu(
        unsigned char *output_rgba8,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int nNodes, 
        float *data, 
        float *grad,
        QuadNodeD *bvh,
        float *disp,
        float *invdet,
        float *tf,
        float2 trans, 
        float stepsize)
{
  fprintf(stderr, "[volren] CPU rendering, width=%d, height=%d\n", viewport[2], viewport[3]);
#pragma omp parallel for collapse(2)
  for (uint x = 0; x < viewport[2]; x ++) {
    for (uint y = 0; y < viewport[3]; y ++) {
      float3 rayO, rayD;
      setup_ray(viewport, invmvp, x, y, rayO, rayD);

      float4 dst = make_float4(0.f); 
      raycasting<SHADING>(dst, nPhi, nNodes, data, grad, bvh, disp, invdet, tf, trans, rayO, rayD, stepsize);

      output_rgba8[(y*viewport[2]+x)*4+0] = clamp(dst.x, 0.f, 1.f) * 255;
      output_rgba8[(y*viewport[2]+x)*4+1] = clamp(dst.y, 0.f, 1.f) * 255;
      output_rgba8[(y*viewport[2]+x)*4+2] = clamp(dst.z, 0.f, 1.f) * 255;
      output_rgba8[(y*viewport[2]+x)*4+3] = clamp(dst.w, 0.f, 1.f) * 255;
    }
  }
}

/////////////////////////////
extern "C" {

void rc_test_point_locator(ctx_rc *ctx, float x, float y)
{
#if WITH_CUDA
  test_point_locator_kernel<<<1, 1>>>(
      (int*)ctx->d_output_rgba8, 
      x, y, ctx->d_bvh, ctx->d_invdet);

  int nid;
  hipMemcpy(&nid, ctx->d_output_rgba8, sizeof(int), hipMemcpyDeviceToHost);
  fprintf(stderr, "[rc_test_point_locator] x={%f, %f}, nid=%d\n", x, y, nid);
#endif
}

void rc_render(ctx_rc *ctx)
{
#if WITH_CUDA
  const dim3 blockSize(16, 16); 
  const dim3 gridSize = dim3(iDivUp(ctx->viewport[2], blockSize.x), iDivUp(ctx->viewport[3], blockSize.y));

  hipMemcpy(ctx->d_viewport, ctx->viewport, sizeof(int)*4, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_viewport), ctx->viewport, sizeof(int)*4);
  hipMemcpy(ctx->d_invmvp, ctx->invmvp, sizeof(float)*16, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_invmvp), ctx->invmvp, sizeof(float)*16);
 
  raycasting_kernel<0><<<gridSize, blockSize>>>(
          ctx->d_output_rgba8,
          ctx->d_viewport, 
          ctx->d_invmvp,
          ctx->nPhi, 
          ctx->nNodes,
          ctx->d_data, 
          ctx->d_grad,
          ctx->d_bvh,
          ctx->d_disp,
          ctx->d_invdet,
          ctx->d_tf,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);

  hipDeviceSynchronize();
  checkLastCudaError("[rc_render]");
#else
  rc_render_cpu(ctx);
#endif
}

void rc_render_cpu(ctx_rc *ctx)
{
  raycasting_cpu<0>(
          (unsigned char*)ctx->h_output,
          ctx->viewport, 
          ctx->invmvp,
          ctx->nPhi, 
          ctx->nNodes,
          ctx->h_data, 
          ctx->h_grad,
          ctx->h_bvh,
          ctx->h_disp,
          ctx->h_invdet,
          ctx->h_tf,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);
}

void rc_bind_bvh(ctx_rc *ctx, int nQuadNodes, QuadNodeD *bvh)
{
  ctx->h_bvh = bvh;
#if WITH_CUDA
  if (ctx->d_bvh != NULL)
    hipFree(ctx->d_bvh);

  hipMalloc((void**)&ctx->d_bvh, sizeof(QuadNodeD)*nQuadNodes);
  hipMemcpy(ctx->d_bvh, bvh, sizeof(QuadNodeD)*nQuadNodes, hipMemcpyHostToDevice);
#endif
}

void rc_set_default_tf(ctx_rc *ctx)
{
  float r[3] = {0.7215686274509804, 0.1803921568627451, 0.1803921568627451}, 
        b[3] = {0.2, 0.4, 0.8};

  float *tf = ctx->h_tf;
  for (int i=0; i<size_tf; i++) {
    float x = (float)i / (size_tf-1);
    tf[i*4] = x*b[0] + (1-x)*r[0];
    tf[i*4+1] = x*b[1] + (1-x)*r[1];
    tf[i*4+2] = x*b[2] + (1-x)*r[2];
    tf[i*4+3] = fmin(0.999f, (x-0.5)*(x-0.5)*40);
  }
  rc_set_tf(ctx, tf);
}

void rc_set_tf(ctx_rc *ctx, float *tf) 
{
  if (tf != ctx->h_tf)
    memcpy(ctx->h_tf, tf, sizeof(float)*size_tf*4);

#if WITH_CUDA
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

  hipMemcpy(ctx->d_tf, tf, sizeof(float)*size_tf*4, hipMemcpyHostToDevice);
  hipMemcpyToArray( ctx->d_tfArray, 0, 0, tf, size_tf*4, hipMemcpyHostToDevice ); 

  texTransferFunc.normalized = true; 
  texTransferFunc.filterMode = hipFilterModeLinear; 
  texTransferFunc.addressMode[0] = hipAddressModeClamp; 
  hipBindTextureToArray(texTransferFunc, ctx->d_tfArray, channelDesc); 

  checkLastCudaError("[rc_set_tf]");
#endif
}

void rc_bind_disp(ctx_rc *ctx, int nNodes, float *disp)
{
  ctx->h_disp = disp;
#if WITH_CUDA
  if (ctx->d_disp == NULL)
    hipMalloc((void**)&ctx->d_disp, sizeof(float)*nNodes*2);
  hipMemcpy(ctx->d_disp, disp, sizeof(float)*nNodes*2, hipMemcpyHostToDevice);
#endif
}

void rc_bind_invdet(ctx_rc *ctx, int nTriangles, float *invdet)
{
  ctx->h_invdet = invdet;
#if WITH_CUDA
  if (ctx->d_invdet == NULL)
    hipMalloc((void**)&ctx->d_invdet, sizeof(float)*nTriangles);
  hipMemcpy(ctx->d_invdet, invdet, sizeof(float)*nTriangles, hipMemcpyHostToDevice);
#endif
}

void rc_bind_data(ctx_rc *ctx, int nNodes, int nTriangles, int nPhi, float *data, float *grad)
{
  ctx->h_data = data;
  ctx->h_grad = grad;
  ctx->nNodes = nNodes;
  ctx->nPhi = nPhi;
#if WITH_CUDA
  if (ctx->d_data == NULL)
    hipMalloc((void**)&ctx->d_data, sizeof(float)*nNodes*nPhi);
  hipMemcpy(ctx->d_data, data, sizeof(float)*nNodes*nPhi, hipMemcpyHostToDevice);
  if (ctx->d_grad == NULL)
    hipMalloc((void**)&ctx->d_grad, sizeof(float)*nNodes*nTriangles*2);
  hipMemcpy(ctx->d_grad, grad, sizeof(float)*nNodes*nTriangles*2, hipMemcpyHostToDevice);
#endif
}

void rc_create_ctx(ctx_rc **ctx)
{
  *ctx = (ctx_rc*)malloc(sizeof(ctx_rc));
  memset(*ctx, 0, sizeof(ctx_rc));

  (*ctx)->h_tf = (float*)malloc(sizeof(float)*size_tf*4);

  const size_t max_npx = 4096*4096;
  (*ctx)->h_output = malloc(4*max_npx);

#if WITH_CUDA
  hipSetDevice(0);
  hipMalloc((void**)&((*ctx)->d_output_rgba8), 4*max_npx); 
  
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  hipMallocArray( &(*ctx)->d_tfArray, &channelDesc, size_tf*4, 1 ); 

  hipMalloc((void**)&((*ctx)->d_tf), sizeof(float)*size_tf*4);
  hipMalloc((void**)&((*ctx)->d_viewport), sizeof(int)*4);
  hipMalloc((void**)&((*ctx)->d_invmvp), sizeof(float)*16);
  
  checkLastCudaError("[rc_init]");
#endif
}

void rc_destroy_ctx(ctx_rc **ctx)
{
  // TODO: free any resources
#if WITH_CUDA
  hipFree((*ctx)->d_output_rgba8);
  hipFree((*ctx)->d_disp);
  hipFree((*ctx)->d_invdet);
#endif
  free((*ctx)->h_output);
  free((*ctx)->h_tf);
  free(*ctx); 
  *ctx = NULL; 
}

void rc_set_viewport(ctx_rc *ctx, int x, int y, int w, int h)
{
  ctx->viewport[0] = x; 
  ctx->viewport[1] = y; 
  ctx->viewport[2] = w; 
  ctx->viewport[3] = h; 
}

void rc_set_range(ctx_rc *ctx, float a, float b)
{
  float c = 1.f/(b-a);
  ctx->trans[0] = c; 
  ctx->trans[1] = -a*c; 
}

void rc_set_stepsize(ctx_rc *ctx, float stepsize)
{
  ctx->stepsize = stepsize;
}

void rc_set_invmvpf(ctx_rc *ctx, float *invmvp)
{
  memcpy(ctx->invmvp, invmvp, sizeof(float)*16); 
}

void rc_set_invmvpd(ctx_rc *ctx, double *invmvp)
{
  for (int i=0; i<16; i++) {
    ctx->invmvp[i] = invmvp[i]; 
  }
}

void rc_clear_output(ctx_rc *ctx)
{
  memset(ctx->h_output, 0, ctx->viewport[2]*ctx->viewport[3]*4);
#if WITH_CUDA
  hipMemset(ctx->d_output_rgba8, 0, 4*ctx->viewport[2]*ctx->viewport[3]);
#endif
}

void rc_copy_output_to_host(ctx_rc *ctx)
{
#if WITH_CUDA
  hipMemcpy(ctx->h_output, ctx->d_output_rgba8, 4*ctx->viewport[2]*ctx->viewport[3], hipMemcpyDeviceToHost);
#if 0
  unsigned char* img = (unsigned char*)ctx->h_output;
  const int j = 0;
  for (int i=0; i<720; i++) {
    fprintf(stderr, "%f\n", img[(j*720+i)*4+3]);
  }
#endif
#endif
}

} // extern "C" 
/////////////////

#include "hip/hip_runtime.h"
#include "volren.cuh"
#include "bvh.cuh"
#include "common.cuh"

__device__ __host__
bool QuadNodeD_insideQuad(const QuadNodeD &q, float x, float y)
{
  return x >= q.Ax && x < q.Bx && y >= q.Ay && y < q.By;
}

__device__ __host__
bool QuadNodeD_insideTriangle(const QuadNodeD &q, float x, float y, float3 &lambda)
{
  lambda.x = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) /
          ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  lambda.y = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) /
         ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  lambda.z = 1.0 - lambda.x - lambda.y;
  // fprintf(stderr, "barycentric=%f, %f, %f\n", lambda.x, lambda.y, lambda.z);
  return lambda.x >= 0 && lambda.y >= 0 && lambda.z >= 0;
}

__device__ __host__
int QuadNodeD_locatePoint_recursive(const QuadNodeD *q, const QuadNodeD *nodes, float x, float y, float3 &lambda)
{
  if (q->triangleId >= 0) { //leaf node
    bool succ = QuadNodeD_insideTriangle(*q, x, y, lambda);
    if (succ) return q->triangleId;
  } else if (QuadNodeD_insideQuad(*q, x, y)) {
    for (int j=0; j<4; j++) {
      if (q->childrenIds[j] > 0) {
        int result = QuadNodeD_locatePoint_recursive(&nodes[q->childrenIds[j]], nodes, x, y, lambda);
        if (result >= 0) return result;
      }
    }
  }
  return -1;
}

__device__ __host__
int QuadNodeD_locatePoint(QuadNodeD *nodes, float x, float y, float3 &lambda)
{
  // float lambda.x, lambda.y, lambda.z;
  static const int maxStackSize = 64;
  int stack[maxStackSize];
  int stackPos = 0;
  stack[stackPos++] = 0; // push root

  while (stackPos > 0) {
    const int i = stack[--stackPos]; // pop
    const QuadNodeD &q = nodes[i];

    // fprintf(stderr, "D_checking node %d, %f, %f, %f, %f\n", i, q.Ax, q.Ay, q.Bx, q.By);
    // fprintf(stderr, "D_checking node %d\n", i);

    if (q.triangleId >= 0) { // leaf node
      bool succ = QuadNodeD_insideTriangle(q, x, y, lambda);
      if (succ) return q.triangleId;
    } else if (QuadNodeD_insideQuad(q, x, y)) { // non-leaf node
      for (int j=0; j<4; j++) {
        if (q.childrenIds[j] > 0)
          stack[stackPos++] = q.childrenIds[j];
      }
    }
  }
  return -1;
}

__device__ __host__
float QuadNodeD_sample(QuadNodeD* bvh, int nid, float3 lambda, float *data) {
  const QuadNodeD &q = bvh[nid];

  return lambda.x * data[q.i0] 
    + lambda.y * data[q.i1]
    + lambda.z * data[q.i2];
}

texture<float4, 1, hipReadModeElementType> texTransferFunc;

// __constant__ int c_viewport[4];
// __constant__ float c_invmvp[16]; 

__device__ __host__
float interpolateXGC(QuadNodeD *bvh, float3 pos, float *data)
{
  // compute cylindrical coordiates
  float r = sqrt(pos.x*pos.x + pos.y + pos.y);
  float theta = atan2(pos.y, pos.x);
  // float z = pos.z;
  
  return 0;
}

template <int SHADING>
__device__ static void rc(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int nNodes,               // number of nodes 
        float *data,              // volume data in unstructured mesh
        QuadNodeD *bvh,
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize, 
        float tnear, float tfar)
{
  const float pi = 3.141592654f;
  const float pi2 = 2*pi;
  float4 src;
  // float3 N, L = make_float3(1, 0, 0), V = rayD; 
  // float3 Ka = make_float3(0.04), 
  //        Kd = make_float3(0.3), 
  //        Ks = make_float3(0.2); 
  // const float delta = 0.5f / dsz.x;   // for shading 
  float3 pos;
  float t = tnear;

  int last_nid = -1, nid;

  while (t < tfar) {
    pos = rayO + rayD*t;

    // cylindar coordinates
    float r = pos.x*pos.x + pos.y*pos.y;
    float phi = atan2(pos.y, pos.x) + pi;
    float z = pos.z;

    float3 lambda;
#if 0 // coherent point locating
    if (last_nid > 0 && QuadNodeD_insideTriangle(bvh[last_nid], r, z, lambda)) // coherent
      nid = last_nid;
    else 
      nid = QuadNodeD_locatePoint(bvh, r, z, lambda);
#else  
    nid = QuadNodeD_locatePoint(bvh, r, z, lambda);
#endif

    if (nid != -1) {
      const float unitAngle = pi2/nPhi;

      int p0 = (int)(phi/unitAngle)%nPhi;
      int p1 = (p0+1)%nPhi;
    
      float alpha = (phi - unitAngle*p0) / unitAngle;
      float v0 = QuadNodeD_sample(bvh, nid, lambda, data + nNodes*p0); //  + nNodes*p0);
      float v1 = QuadNodeD_sample(bvh, nid, lambda, data + nNodes*p1); //  + nNodes*p1);

      float value = (1-alpha)*v0 + alpha*v1;
      
      float v = clamp(value*0.01, -0.5f, 0.5f);
      // sample = interpolateXGC(bvh, pos, data); 
      // sample = QuadNodeD_sample(bvh, x, y, data);
      // sample = tex3Dtrans<DataType, readMode, TRANSFORM>(texVolume, trans, coords); 
      // src = tex1D(texTransferFunc, sample);
      // src = make_float4(sample, 1.0-sample, 0.0, 0.9);
      // sample = pow(1.f - sample, 2.f); 
      // src = make_float4(sample*2, 1.f-sample*2, 0.0, sample*0.4); 
      // src = make_float4(lambda.x, lambda.y, lambda.z, 0.5);

      src = make_float4(v+0.5, 0.5-v, 0, min(1.f, v*v*10));
      // src = make_float4(phi/(pi*2), 1-phi/(pi*2), 0, 0.3);
      // src = make_float4(p1/8.0, 0.5, 0, 0.3);

#if 0
      if (SHADING) {
        float3 lit; 
        N = gradient(texVolume, coords, delta); 
        lit = cook(N, V, L, Ka, Kd, Ks); 
        src.x += lit.x; 
        src.y += lit.y; 
        src.z += lit.z; 
      }
#endif
      
      src.w = 1.f - pow(1.f - src.w, stepsize); // alpha correction  

      dst.x += (1.0 - dst.w) * src.x * src.w;
      dst.y += (1.0 - dst.w) * src.y * src.w;
      dst.z += (1.0 - dst.w) * src.z * src.w;
      dst.w += (1.0 - dst.w) * src.w;
    }
    
    t += stepsize; 
  }
  // dst.x = 1; dst.y = 0; dst.z = 0; dst.w = 1;
}

template <int SHADING>
__device__ static void raycasting(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int nNodes,               // number of nodes 
        float *data,              // volume data in unstructured mesh
        QuadNodeD *bvh,
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize)           // stepsize
{
  const QuadNodeD &root = bvh[0];
  const float innerRadius = root.Ax, outerRadius = root.Bx;
  // const float innerRadius = 1.0, outerRadius = 1.2;
  const float z0 = root.Ay, z1 = root.By; 
  float tnear0, tfar0, tnear1, tfar1;
  bool b0 = intersectCylinder(rayO, rayD, tnear0, tfar0, outerRadius, z0, z1), 
       b1 = intersectCylinder(rayO, rayD, tnear1, tfar1, innerRadius, z0, z1);
  
#if 1
  if (b0 && (!b1))
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, trans, rayO, rayD, stepsize, tnear0, tfar0);
  else if (b0 && b1) {
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, trans, rayO, rayD, stepsize, tnear0, tnear1);
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, trans, rayO, rayD, stepsize, tfar1, tfar0);
  }
#else
  if (b1)
    rc<SHADING>(dst, nPhi, nNodes, data, bvh, trans, rayO, rayD, stepsize, tnear1, tfar1);
#endif
}

template <int SHADING>
__global__ static void raycasting_kernel(
        float *output,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int nNode, 
        float *data, 
        QuadNodeD *bvh,
        float2 trans, 
        float stepsize)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= viewport[2] || y>= viewport[3]) return;
  
  float coord[4], obj0[4], obj1[4]; 
  coord[0] = (x-viewport[0])*2.f / viewport[2] - 1.f; 
  coord[1] = (y-viewport[1])*2.f / viewport[3] - 1.f; 
  coord[2] = -1.0; 
  coord[3] = 1.0;

  mulmatvec(invmvp, coord, obj0); 
  coord[2] = 1.0; 
  mulmatvec(invmvp, coord, obj1); 
  if (obj0[3] == 0.f || obj1[3] == 0.f) return; 

  for (int i=0; i<3; i++)
      obj0[i] /= obj0[3], obj1[i] /= obj1[3]; 

  float3 rayO = make_float3(obj0[0], obj0[1], obj0[2]), 
         rayD = normalize(make_float3(obj1[0]-obj0[0], obj1[1]-obj0[1], obj1[2]-obj0[2]));
  float4 dst = make_float4(0.f); 

#if 1
  raycasting<SHADING>(dst, nPhi, nNode, data, bvh, trans, rayO, rayD, stepsize);
#else
  dst.x = 1; // rayD.x;
  dst.y = 0; // rayD.y;
  dst.z = 0; // rayD.z;
  dst.w = 1;
#endif

  // GL_ONE_MINUS_DST_ALPHA, GL_ONE
  float w0 = 1-output[(y*viewport[2]+x)*4+3]; //, w1 = 1; make the compiler happy :)

  output[(y*viewport[2]+x)*4+0] += w0* dst.x;
  output[(y*viewport[2]+x)*4+1] += w0* dst.y;
  output[(y*viewport[2]+x)*4+2] += w0* dst.z;
  output[(y*viewport[2]+x)*4+3] += w0* dst.w;
}


/////////////////////////////
extern "C" {

void rc_render(ctx_rc *ctx)
{
  const dim3 blockSize(16, 16); 
  const dim3 gridSize = dim3(iDivUp(ctx->viewport[2], blockSize.x), iDivUp(ctx->viewport[3], blockSize.y));

  hipMemcpy(ctx->d_viewport, ctx->viewport, sizeof(int)*4, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_viewport), ctx->viewport, sizeof(int)*4);
  hipMemcpy(ctx->d_invmvp, ctx->invmvp, sizeof(float)*16, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_invmvp), ctx->invmvp, sizeof(float)*16);
 
  raycasting_kernel<0><<<gridSize, blockSize>>>(
          ctx->d_output,
          ctx->d_viewport, 
          ctx->d_invmvp,
          ctx->nPhi, 
          ctx->nNodes,
          ctx->d_data, 
          ctx->d_bvh,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);
  checkLastCudaError("[rc_render]");
}

void rc_bind_bvh(ctx_rc *ctx, int nQuadNodes, QuadNodeD *bvh)
{
  if (ctx->d_bvh != NULL)
    hipFree(ctx->d_bvh);

  hipMalloc((void**)&ctx->d_bvh, sizeof(QuadNodeD)*nQuadNodes);
  hipMemcpy(ctx->d_bvh, bvh, sizeof(QuadNodeD)*nQuadNodes, hipMemcpyHostToDevice);
}

void rc_bind_transfer_function_array(hipArray* array)
{
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>(); 

  texTransferFunc.normalized = true; 
  texTransferFunc.filterMode = hipFilterModeLinear; 
  texTransferFunc.addressMode[0] = hipAddressModeClamp; 
  hipBindTextureToArray(texTransferFunc, array, channelDesc); 

  checkLastCudaError("[rc_bind_transfer_function_array]");
}

void rc_bind_data(ctx_rc *ctx, int nNodes, int nPhi, const float *data)
{
  ctx->nNodes = nNodes;
  ctx->nPhi = nPhi;
  if (ctx->d_data == NULL)
    hipMalloc((void**)&ctx->d_data, sizeof(float)*nNodes*nPhi);
  hipMemcpy(ctx->d_data, data, sizeof(float)*nNodes*nPhi, hipMemcpyHostToDevice);
}

void rc_create_ctx(ctx_rc **ctx)
{
  hipSetDevice(0);

  *ctx = (ctx_rc*)malloc(sizeof(ctx_rc));
  memset(*ctx, 0, sizeof(ctx_rc));

  const size_t max_npx = 4096*4096;

  hipMalloc((void**)&((*ctx)->d_output), sizeof(float)*max_npx); 
  (*ctx)->h_output = malloc(sizeof(float)*max_npx);

  hipMalloc((void**)&((*ctx)->d_viewport), sizeof(int)*4);
  hipMalloc((void**)&((*ctx)->d_invmvp), sizeof(float)*16);
  
  checkLastCudaError("[rc_init]");
}

void rc_destroy_ctx(ctx_rc **ctx)
{
  // TODO: free any resources
  // hipFree(ctx->d_output);
  free((*ctx)->h_output);
  free(*ctx); 
  *ctx = NULL; 
}

void rc_set_viewport(ctx_rc *ctx, int x, int y, int w, int h)
{
  ctx->viewport[0] = x; 
  ctx->viewport[1] = y; 
  ctx->viewport[2] = w; 
  ctx->viewport[3] = h; 
}

void rc_set_range(ctx_rc *ctx, float a, float b)
{
  float c = 1.f/(b-a);
  ctx->trans[0] = c; 
  ctx->trans[1] = -a*c; 
}

void rc_set_stepsize(ctx_rc *ctx, float stepsize)
{
  ctx->stepsize = stepsize;
}

void rc_set_invmvpf(ctx_rc *ctx, float *invmvp)
{
  memcpy(ctx->invmvp, invmvp, sizeof(float)*16); 
}

void rc_set_invmvpd(ctx_rc *ctx, double *invmvp)
{
  for (int i=0; i<16; i++) {
    ctx->invmvp[i] = invmvp[i]; 
  }
}

void rc_clear_output(ctx_rc *ctx)
{
  hipMemset(ctx->d_output, 0, 4*sizeof(float)*ctx->viewport[2]*ctx->viewport[3]);
}

void rc_copy_output_to_host(ctx_rc *ctx)
{
  hipMemcpy(ctx->h_output, ctx->d_output, 4*sizeof(float)*ctx->viewport[2]*ctx->viewport[3], hipMemcpyDeviceToHost); 
}

void rc_copy_output_to_host_rgb8(ctx_rc *ctx)
{
  const size_t npx = ctx->viewport[2] * ctx->viewport[3];
  hipMemcpy(ctx->h_output, ctx->d_output, 4*sizeof(float)*npx, hipMemcpyDeviceToHost);
  float *ffb = (float*)ctx->h_output;
  unsigned char *ufb = (unsigned char*)ctx->h_output;

  for (int i=0; i<npx; i++) {
    ufb[i*3] = ffb[i*4]*255;
    ufb[i*3+1] = ffb[i*4+1]*255;
    ufb[i*3+2] = ffb[i*4+2]*255;
  }
}

} // extern "C" 
/////////////////

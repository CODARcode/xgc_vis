#include "hip/hip_runtime.h"
#include "volren.cuh"
#include "bvh.cuh"
#include "mu.cuh"
#include "common.cuh"

#define BVH_NUM_CHILDREN 4
#define PREINT 0

static void createPreIntegrationTable(float *lut_preint, float *lut, int resolution)
{
  static const int MAX_RESOLUTION_LUT = 256; // 1024 

	float r=0.f, g=0.f, b=0.f, a=0.f;
	float rcol,  gcol,  bcol,  acol;
	float rInt[MAX_RESOLUTION_LUT],
		    gInt[MAX_RESOLUTION_LUT],
		    bInt[MAX_RESOLUTION_LUT],
		    aInt[MAX_RESOLUTION_LUT];
	int smin, smax;
	float factor, tauc;
	int lookupIndex = 0;
	
	rInt[0] = 0.f; gInt[0] = 0.f, bInt[0] = 0.f, aInt[0] = 0.f;

	// compute integral functions
	for (int i=1; i<resolution; i++) {
		tauc = (lut[(i-1)*4+3] + lut[i*4+3]) / 2.f;
		r   += (lut[(i-1)*4+0] + lut[i*4+0]) / 2.f;// * tauc;// / resolution;
		g   += (lut[(i-1)*4+1] + lut[i*4+1]) / 2.f;// * tauc;// / resolution;
		b   += (lut[(i-1)*4+2] + lut[i*4+2]) / 2.f;// * tauc;// / resolution;
		a   += tauc;

		rInt[i] = r; gInt[i] = g; bInt[i] = b; aInt[i] = a;
	}

	// compute look-up table from integral functions
	for (int sb=0; sb<resolution; sb++) {
		for (int sf=0; sf<resolution; sf++) {
			if (sb < sf) {smin=sb; smax = sf;}
			else {smin=sf; smax=sb;}

			if (smax != smin) {
				factor = 1.f / (float)(smax - smin);
				rcol   = (rInt[smax] - rInt[smin]) * factor;
				gcol   = (gInt[smax] - gInt[smin]) * factor;
				bcol   = (bInt[smax] - bInt[smin]) * factor;
				acol   = 1.f - exp(-(aInt[smax] - aInt[smin]) * factor);
			} else {
				factor = 1.f;
				rcol   = lut[smin*4+0] * lut[smin*4+3] * factor;
				gcol   = lut[smin*4+1] * lut[smin*4+3] * factor;
				bcol   = lut[smin+4+2] * lut[smin*4+3] * factor;
				acol   = (1.f - exp(-lut[smin*4+3]) * factor );
			}

			lut_preint[lookupIndex++] = clamp(rcol, 0.f, 1.f);
			lut_preint[lookupIndex++] = clamp(gcol, 0.f, 1.f);
			lut_preint[lookupIndex++] = clamp(bcol, 0.f, 1.f);
			lut_preint[lookupIndex++] = clamp(acol, 0.f, 1.f);
		}
  }
}


__device__ __host__
inline bool BVHNodeD_insideQuad(const BVHNodeD &q, float x, float y)
{
#if __CUDA_ARCH__
  return __fmul_rz(x-q.Ax, x-q.Bx) <= 0 && __fmul_rz(y-q.Ay, y-q.By) <= 0;
#else
  return (x - q.Ax) * (x - q.Bx) <= 0 && (y - q.Ay) * (y - q.By) <= 0;
#endif
  // return x >= q.Ax && x < q.Bx && y >= q.Ay && y < q.By;
}

__device__ __host__
inline bool BVHNodeD_insideTriangle(const BVHNodeD &q, float x, float y, float3 &lambda, float *invdet)
{
#if 0
  lambda.x = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) /
          ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  lambda.y = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) /
         ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
#endif
  const float d = invdet[q.triangleId];
  lambda.x = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) * d; 
  lambda.y = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) * d;
  lambda.z = 1.0 - lambda.x - lambda.y;
  // fprintf(stderr, "barycentric=%f, %f, %f\n", lambda.x, lambda.y, lambda.z);
  return lambda.x >= 0 && lambda.y >= 0 && lambda.z >= 0;
}

__device__ __host__
inline int BVHNodeD_locatePoint_recursive(const BVHNodeD *q, const BVHNodeD *nodes, float x, float y, float3 &lambda, float *invdet)
{
  if (q->triangleId >= 0) { //leaf node
    bool succ = BVHNodeD_insideTriangle(*q, x, y, lambda, invdet);
    if (succ) return q->triangleId;
  } else if (BVHNodeD_insideQuad(*q, x, y)) {
    for (int j=0; j<BVH_NUM_CHILDREN; j++) {
      if (q->childrenIds[j] > 0) {
        int result = BVHNodeD_locatePoint_recursive(&nodes[q->childrenIds[j]], nodes, x, y, lambda, invdet);
        if (result >= 0) return result;
      }
    }
  }
  return -1;
}

__device__ __host__
inline int BVHNodeD_locatePoint(BVHNodeD *nodes, float x, float y, float3 &lambda, float *invdet, int root=0)
{
  // float lambda.x, lambda.y, lambda.z;
  static const int maxStackSize = 64;
  int stack[maxStackSize];
  int stackPos = 0;
  stack[stackPos++] = root; // push root

  while (stackPos > 0) {
    const int i = stack[--stackPos]; // pop
    const BVHNodeD &q = nodes[i];

    // fprintf(stderr, "D_checking node %d, %f, %f, %f, %f\n", i, q.Ax, q.Ay, q.Bx, q.By);
    // fprintf(stderr, "D_checking node %d\n", i);

    if (q.triangleId >= 0) { // leaf node
      bool succ = BVHNodeD_insideTriangle(q, x, y, lambda, invdet);
      if (succ) return i; // q.triangleId;
    } else if (BVHNodeD_insideQuad(q, x, y)) { // non-leaf node
      for (int j=0; j<BVH_NUM_CHILDREN; j++) {
        if (q.childrenIds[j] > 0)
          stack[stackPos++] = q.childrenIds[j];
      }
    }
  }
  return -1;
}

__device__ __host__
inline int BVHNodeD_locatePoint_coherent(BVHNodeD *bvh, int last_nid, float x, float y, float3 &lambda, float *invdet, int *neighbors)
{
  // check if last_nid is valid
  if (last_nid<0) return BVHNodeD_locatePoint(bvh, x, y, lambda, invdet);

  // check if in the same triangle
  if (BVHNodeD_insideTriangle(bvh[last_nid], x, y, lambda, invdet)) return last_nid;

  // check if neighbor triangles have the point
#if 0
  for (int i=0; i<3; i++) {
    int triangleId = bvh[last_nid].triangleId;
    int neighborQuadId = neighbors[triangleId*3];
    if (neighborQuadId<0) continue;
    else if (BVHNodeD_insideTriangle(bvh[neighborQuadId], x, y, lambda, invdet)) return neighborQuadId;
  }
#endif

  // traverse from parents
  // int nid = BVHNodeD_locatePoint(bvh, x, y, lambda, invdet, bvh[bvh[last_nid].parentId].parentId);
  // int nid = BVHNodeD_locatePoint(bvh, x, y, lambda, invdet, bvh[last_nid].parentId);
  // if (nid >= 0) return nid;

  // TODO: check if in triangle neighbors of last_nid

  // fallback
  return BVHNodeD_locatePoint(bvh, x, y, lambda, invdet);
}

__device__ __host__
inline float BVHNodeD_sample(int i0, int i1, int i2, float3 lambda, float *data) {
  return lambda.x * data[i0] + lambda.y * data[i1] + lambda.z * data[i2];
}

__device__ __host__
inline float2 BVHNodeD_sample2(int i0, int i1, int i2, float3 lambda, float *data) {
  return make_float2(lambda.x * data[i0*2] + lambda.y * data[i1*2] + lambda.z * data[i2*2],
      lambda.x * data[i0*2+1] + lambda.y * data[i1*2+1] + lambda.z * data[i2*2+1]);
}

__device__ __host__
inline float BVHNodeD_sample(BVHNodeD* bvh, int nid, float3 lambda, float *data) {
  const BVHNodeD &q = bvh[nid];
  return lambda.x * data[q.i0] + lambda.y * data[q.i1] + lambda.z * data[q.i2];
}

__device__ __host__
inline float2 BVHNodeD_sample2(BVHNodeD* bvh, int nid, float3 lambda, float *data) {
  const BVHNodeD &q = bvh[nid];
  return BVHNodeD_sample2(q.i0, q.i1, q.i2, lambda, data);
  // return make_float2(lambda.x * data[q.i0*2] + lambda.y * data[q.i1*2] + lambda.z * data[q.i2*2],
  //     lambda.x * data[q.i0*2+1] + lambda.y * data[q.i1*2+1] + lambda.z * data[q.i2*2+1]);
}

template <int PSI, int SHADING>
__device__ __host__
inline int interpolateXGC(float &value, float3 &g, int &last_nid, BVHNodeD *bvh,
    float3 p, float r2, float r, float phi, float z, float &alpha,
    float2 psi_range, float2 angle_range, int nPhi, int iPhi, int nNodes, int nTriangles, float *data, float2 *grad, float *invdet, int *neighbors, float *psi)
{
  static const float pi = 3.141592654f;
  static const float pi2 = 2*pi;
  
  float3 lambda;
  int nid = BVHNodeD_locatePoint_coherent(bvh, last_nid, r, z, lambda, invdet, neighbors);
  if (nid == -1) return nid;
  last_nid = nid;
 
  const BVHNodeD &q = bvh[nid];
  
  if (PSI) {
    float psi_val = BVHNodeD_sample(q.i0, q.i1, q.i2, lambda, psi);
    // if (psi_val > 0.1) return -1;
    if ((psi_val-psi_range.x)*(psi_val-psi_range.y) > 0) // outside psi_range
      return -1;
  }

  const float deltaAngle = pi2/(nPhi*iPhi);
#ifdef __CUDA_ARCH__
  int p0 = __float2int_rd(__fdividef(phi, deltaAngle)) % nPhi;
#else
  int p0 = (int)(phi/deltaAngle) % nPhi;
#endif
  int p1 = (p0+1)%nPhi;

  // coef
  // float alpha = __fdiv_rd((phi - __fmul_rd(deltaAngle, p0)), deltaAngle);
  alpha = (phi - deltaAngle*p0) / deltaAngle;

  // value interpolation
  // float v0 = BVHNodeD_sample(bvh, nid, lambda, data + nNodes*p0);
  // float v1 = BVHNodeD_sample(bvh, nid, lambda, data + nNodes*p1);
  float v0 = BVHNodeD_sample(q.i0, q.i1, q.i2, lambda, data + nNodes*p0);
  float v1 = BVHNodeD_sample(q.i0, q.i1, q.i2, lambda, data + nNodes*p1);
  value = (1-alpha)*v0 + alpha*v1;

  if (SHADING) {
    // gradient interpolation
    float2 cgrad0 = grad[nTriangles*p0 + q.triangleId];
    float2 cgrad1 = grad[nTriangles*p1 + q.triangleId];
    // float2 cgrad0 = make_float2(grad[(nTriangles*p0 + q.triangleId)*2], grad[(nTriangles*p0 + q.triangleId)*2+1]);
    // float2 cgrad1 = make_float2(grad[(nTriangles*p1 + q.triangleId)*2], grad[(nTriangles*p1 + q.triangleId)*2+1]);
    float2 cgrad = (1-alpha)*cgrad0 + alpha*cgrad1;
    g = make_float3(p.x/r*cgrad.x - p.y/r2, p.y/r*cgrad.x - p.x/r2, cgrad.y);
  }

  return nid;
}


template <int PSI, int SHADING>
__device__ __host__
inline int interpolateXGC2(float &value, float3 &g, int &last_nid, BVHNodeD *bvh,
    float3 p, float r2, float r, float phi, float z, float &alpha,
    float2 psi_range, float2 angle_range, int nPhi, int iPhi, int nNodes, int nTriangles, float *data, float2 *grad, float *invdet, int *neighbors, float *psi, float *disp)
{
  static const float pi = 3.141592654f;
  static const float pi2 = 2*pi;
  
  float3 lambda;
  int nid = BVHNodeD_locatePoint_coherent(bvh, last_nid, r, z, lambda, invdet, neighbors);
  if (nid == -1) return nid; 
  last_nid = nid;
      
  const float deltaAngle = pi2/nPhi;

  int p0 = (int)(phi/deltaAngle)%nPhi;
  int p1 = (p0+1)%nPhi;
  alpha = (phi - deltaAngle*p0) / deltaAngle;

  // interpolate disp
  const BVHNodeD &q = bvh[nid];
  float dx = lambda.x * disp[q.i0*2] + lambda.y * disp[q.i1*2] + lambda.z * disp[q.i2*2];
  float dy = lambda.x * disp[q.i0*2+1] + lambda.y * disp[q.i1*2+1] + lambda.z * disp[q.i2*2+1];
 
  float3 lambda0, lambda1;
  int nid0 = BVHNodeD_locatePoint_coherent(bvh, nid, r+dx*(1-alpha), z+dy*(1-alpha), lambda0, invdet, neighbors);
  int nid1 = BVHNodeD_locatePoint_coherent(bvh, nid, r+dx*alpha, z+dy*alpha, lambda1, invdet, neighbors);
  if (nid0 == -1 || nid1 == -1) {
    // fprintf(stderr, "nid=%d, nid0=%d, nid1=%d, dx=%f, dy=%f\n", nid, nid0, nid1, dx, dy);
    return -1;
  }

  float v0 = BVHNodeD_sample(bvh, nid0, lambda0, data + nNodes*p0); //  + nNodes*p0);
  float v1 = BVHNodeD_sample(bvh, nid1, lambda1, data + nNodes*p1); //  + nNodes*p1);
  value = (1-alpha)*v0 + alpha*v1;

  // if (alpha<0 || alpha>=1) fprintf(stderr, "%f\n", alpha);

  if (SHADING) {
    // gradient interpolation
    float2 cgrad0 = grad[nTriangles*p0 + q.triangleId];
    float2 cgrad1 = grad[nTriangles*p1 + q.triangleId];
    float2 cgrad = (1-alpha)*cgrad0 + alpha*cgrad1;
    g = make_float3(p.x/r*cgrad.x - p.y/r2, p.y/r*cgrad.x - p.x/r2, cgrad.y);
  }

  return nid;
}

__device__ __host__
static inline float4 value2color_preint(float value, float value1, float4 *ptf, float2 trans) 
{
#ifdef __CUDA_ARCH__
  const float x = __saturatef(value * trans.x + trans.y);
  const float y = __saturatef(value1 * trans.x + trans.y);
#else
  const float x = clamp(value * trans.x + trans.y, 0.f, 1.f);
  const float y = clamp(value1 * trans.x + trans.y, 0.f, 1.f);
#endif
  
  static const int n = 256;
  static const float delta = 1.f / (n-1);
#ifdef __CUDA_ARCH__
  const int i = min(__float2int_rd(x*(n-1)), n-2) , i1 = i + 1;
  const int j = min(__float2int_rd(y*(n-1)), n-2) , j1 = j + 1;
#else
  const int i = min((int)(x*(n-1)), n-2) , i1 = i + 1;
  const int j = min((int)(y*(n-1)), n-2) , j1 = j + 1;
#endif
  const float ibeta = x - i*delta, ialpha = 1 - ibeta;
  const float jbeta = y - j*delta, jalpha = 1 - jbeta;

  float4 c0 = ialpha * ptf[i*n+j] + ibeta * ptf[i1*n+j], 
         c1 = ialpha * ptf[i*n+j1] + ibeta * ptf[i1*n+j1];

  return jalpha * c0 + jbeta * c1;
  // return alpha * tf[i] + beta * tf[i1];
}

__device__ __host__ 
static inline float4 value2color(float value, float4 *tf, float2 trans)
{
#ifdef __CUDA_ARCH__
  const float x = __saturatef(value * trans.x + trans.y);
#else
  const float x = clamp(value * trans.x + trans.y, 0.f, 1.f);
#endif

  // float v = x-0.5;
  // return make_float4(x, 1-x, 0, fminf(0.999f, v*v*40));
  
  static const int n = 256;
  static const float delta = 1.f / (n-1);
#ifdef __CUDA_ARCH__
  const int i = min(__float2int_rd(x*(n-1)), n-2) , j = i + 1;
#else
  const int i = min((int)(x*(n-1)), n-2) , j = i + 1;
#endif
  const float beta = x - i*delta, alpha = 1 - beta;

  return alpha * tf[i] + beta * tf[j];
#if 0
  return make_float4(
      alpha * tf[i*4] + beta * tf[j*4], 
      alpha * tf[i*4+1] + beta * tf[j*4+1], 
      alpha * tf[i*4+2] + beta * tf[j*4+2], 
      alpha * tf[i*4+3] + beta * tf[j*4+3]);
#endif
}

  
template <int ANGLE, int PSI, int SHADING>
__device__ __host__ static inline void rc(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int iPhi, 
        int nNodes,               // number of nodes 
        int nTriangles,           // number of triangles 
        float *data,              // volume data in unstructured mesh
        float2 *grad,              // gradient
        BVHNodeD *bvh,
        float *disp,
        float *invdet,
        int *neighbors,
        float *psi,
        float2 psi_range,
        float2 angle_range,
        float slice_highlight_ratio,
        float3 Ka, 
        float3 Kd, 
        float3 Ks,
        float3 L,
        float4 *tf,
        float4 *ptf,
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize, 
        float tnear, float tfar)
{
  float4 src;
  float3 N, /* L = make_float3(-1, 0, 0),*/  V = rayD; 
  // const float3 Ka = make_float3(0.04), Kd = make_float3(0.3), Ks = make_float3(0.2); 
  float3 p, g; // position and gradient
  float value, value1;
  float t = tnear;
  int nid = -2, nid1, last_nid = -1;

  while (t < tfar) {
    p = rayO + rayD*t;
  
    // cylindar coordinates
#ifdef __CUDA_ARCH__
    float r2 = __fmul_rd(p.x, p.x) + __fmul_rd(p.y, p.y);
    float r = __fsqrt_rd(r2);
#else
    float r2 = p.x*p.x + p.y*p.y;
    float r = sqrt(r2);
#endif
    float phi = atan2(p.y, p.x) + pi;
    float z = p.z;
    
    if (ANGLE && (phi-angle_range.x)*(phi-angle_range.y) > 0) {
      t += stepsize; 
      continue;
    }

    float alpha;
#if PREINT // preintegration
    if (nid == -2) { // first time
      nid = interpolateXGC<PSI, SHADING>(value, g, last_nid, bvh, p, r2, r, phi, z, alpha, psi_range, angle_range, nPhi, iPhi, nNodes, nTriangles, data, grad, invdet, neighbors, psi);
      nid1 = interpolateXGC<PSI, SHADING>(value1, g, last_nid, bvh, p+rayD*stepsize, r2, r, phi, z, alpha, psi_range, angle_range, nPhi, iPhi, nNodes, nTriangles, data, grad, invdet, neighbors, psi);
    } else {
      value = value1;
      nid = nid1;
      nid1 = interpolateXGC<PSI, SHADING>(value1, g, last_nid, bvh, p+rayD*stepsize, r2, r, phi, z, alpha, psi_range, angle_range, nPhi, iPhi, nNodes, nTriangles, data, grad, invdet, neighbors, psi);
    }
#else
    nid = interpolateXGC<PSI, SHADING>(value, g, last_nid, bvh, p, r2, r, phi, z, alpha, psi_range, angle_range, nPhi, iPhi, nNodes, nTriangles, data, grad, invdet, neighbors, psi);
    // nid = interpolateXGC2<PSI, SHADING>(value, g, last_nid, bvh, p, r2, r, phi, z, alpha, psi_range, angle_range, nPhi, iPhi, nNodes, nTriangles, data, grad, invdet, neighbors, psi, disp);
#endif
 
    if (nid >= 0) {
#if PREINT
      src = value2color_preint(value, value1, ptf, trans);
#else
      src = value2color(value, tf, trans);
#endif  
      // if (alpha < 0.001) src.w = fminf(0.999f, src.w*slice_highlight_ratio); // TODO: optimize if
      // if (alpha < 0.01) src.w = 0.999f;

      if (SHADING) {
        float3 lit; 
        N = normalize(g);
        lit = cook(N, V, L, Ka, Kd, Ks);
        // lit = phong(N, V, L, Ka, Kd, Ks, 100);
#ifdef  __CUDA_ARCH__
        // src = make_float4(N.x, N.y, N.z, src.w);
        src.x = __saturatef(src.x + lit.x); 
        src.y = __saturatef(src.y + lit.y); 
        src.z = __saturatef(src.z + lit.z); 
#else
        src.x = clamp(src.x + lit.x, 0.f, 1.f);
        src.y = clamp(src.y + lit.y, 0.f, 1.f);
        src.z = clamp(src.z + lit.z, 0.f, 1.f);
#endif
      }
     
#ifdef __CUDA_ARCH__
      src.w = 1.f - __powf(1.f - src.w, stepsize*4); // alpha correction  
#else
      src.w = 1.f - pow(1.f - src.w, stepsize*4); // alpha correction  
#endif

      dst.x += (1.0 - dst.w) * src.x * src.w;
      dst.y += (1.0 - dst.w) * src.y * src.w;
      dst.z += (1.0 - dst.w) * src.z * src.w;
      dst.w += (1.0 - dst.w) * src.w;
    }
    
    if (dst.w > 0.98) return; // early ray termination
    t += stepsize; 
  }
  // dst.x = 1; dst.y = 0; dst.z = 0; dst.w = 1;
}

template <int ANGLE, int PSI, int SHADING>
__device__ __host__ static inline void raycasting(
        float4 &dst,              // destination color
        int nPhi,                 // number of planes
        int iPhi,
        int nNodes,               // number of nodes 
        int nTriangles, 
        float *data,              // volume data in unstructured mesh
        float2 *grad,
        BVHNodeD *bvh,
        float *disp,
        float *invdet,
        int *neighbors,
        float *psi,
        float2 psi_range,
        float2 angle_range,
        float slice_highlight_ratio,
        float3 Ka, 
        float3 Kd, 
        float3 Ks,
        float3 L,
        float4 *tf, 
        float4 *ptf,
        float2 trans,             // range transformation 
        float3 rayO,              // ray origin 
        float3 rayD,              // ray direction
        float stepsize)           // stepsize
{
  const BVHNodeD &root = bvh[0];
  const float innerRadius = root.Ax, outerRadius = root.Bx;
  const float z0 = root.Ay, z1 = root.By; 
  float tnear0=-FLT_MAX, tfar0=FLT_MAX, tnear1=-FLT_MAX, tfar1=FLT_MAX;
  bool b0 = intersectCylinder(rayO, rayD, tnear0, tfar0, outerRadius, z0, z1), 
       b1 = intersectCylinder(rayO, rayD, tnear1, tfar1, innerRadius, z0, z1);
  
#if 1
  if (b0 && (!b1))
    rc<ANGLE, PSI, SHADING>(dst, nPhi, iPhi, nNodes, nTriangles, data, grad, bvh, disp, invdet, neighbors, psi, psi_range, angle_range, 
        slice_highlight_ratio, Ka, Kd, Ks, L, tf, ptf, trans, rayO, rayD, stepsize, tnear0, tfar0);
  else if (b0 && b1) {
    rc<ANGLE, PSI, SHADING>(dst, nPhi, iPhi, nNodes, nTriangles, data, grad, bvh, disp, invdet, neighbors, psi, psi_range, angle_range, 
        slice_highlight_ratio, Ka, Kd, Ks, L, tf, ptf, trans, rayO, rayD, stepsize, tnear0, tnear1);
    rc<ANGLE, PSI, SHADING>(dst, nPhi, iPhi, nNodes, nTriangles, data, grad, bvh, disp, invdet, neighbors, psi, psi_range, angle_range, 
        slice_highlight_ratio, Ka, Kd, Ks, L, tf, ptf, trans, rayO, rayD, stepsize, tfar1, tfar0);
  }
#else
  if (b0) {
    rc<SHADING>(dst, nPhi, iPhi, nNodes, data, bvh, disp, tf, trans, rayO, rayD, stepsize, tnear0, tfar0);
  }
#endif
}

#if WITH_CUDA
__global__ static void test_point_locator_kernel(
    int *output, 
    float x, float y,
    BVHNodeD *bvh,
    float *invdet)
{
  float3 lambda;
  *output = BVHNodeD_locatePoint(bvh, x, y, lambda, invdet);
}
#endif

__device__ __host__ inline bool setup_ray(
    int *viewport, 
    float *invmvp, 
    uint x, uint y,
    float3 &rayO, float3 &rayD)
{
  float coord[4], obj0[4], obj1[4]; 
  coord[0] = (x-viewport[0])*2.f / viewport[2] - 1.f; 
  coord[1] = (y-viewport[1])*2.f / viewport[3] - 1.f; 
  coord[2] = -1.0; 
  coord[3] = 1.0;

  mulmatvec(invmvp, coord, obj0); 
  coord[2] = 1.0; 
  mulmatvec(invmvp, coord, obj1); 
  if (obj0[3] == 0.f || obj1[3] == 0.f) return false; 

  for (int i=0; i<3; i++)
      obj0[i] /= obj0[3], obj1[i] /= obj1[3]; 

  rayO = make_float3(obj0[0], obj0[1], obj0[2]);
  rayD = normalize(make_float3(obj1[0]-obj0[0], obj1[1]-obj0[1], obj1[2]-obj0[2]));
  return true;
}

#if WITH_CUDA
template <int ANGLE, int PSI, int SHADING>
__global__ static void raycasting_kernel(
        unsigned char *output_rgba8,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int iPhi,
        int nNodes, 
        int nTriangles, 
        float *data, 
        float2 *grad,
        BVHNodeD *bvh,
        float *disp,
        float *invdet,
        int *neighbors,
        float *psi,
        float2 psi_range,
        float2 angle_range,
        float slice_highlight_ratio,
        float3 Ka, 
        float3 Kd, 
        float3 Ks,
        float3 L,
        float4 *tf,
        float4 *ptf,
        float2 trans, 
        float stepsize)
{
  uint x = blockIdx.x*blockDim.x + threadIdx.x;
  uint y = blockIdx.y*blockDim.y + threadIdx.y;

  if (x >= viewport[2] || y>= viewport[3]) return;
  float3 rayO, rayD;
  setup_ray(viewport, invmvp, x, y, rayO, rayD);

  float4 dst = make_float4(0.f); 
  raycasting<ANGLE, PSI, SHADING>(dst, nPhi, iPhi, nNodes, nTriangles, data, grad, bvh, disp, invdet, neighbors, psi, psi_range, angle_range, 
      slice_highlight_ratio, Ka, Kd, Ks, L, tf, ptf, trans, rayO, rayD, stepsize);
  
  output_rgba8[(y*viewport[2]+x)*4+0] = dst.x * 255;
  output_rgba8[(y*viewport[2]+x)*4+1] = dst.y * 255;
  output_rgba8[(y*viewport[2]+x)*4+2] = dst.z * 255;
  output_rgba8[(y*viewport[2]+x)*4+3] = dst.w * 255;

  // if (y == 300) printf("%f\n", dst.w);

#if 0
  // GL_ONE_MINUS_DST_ALPHA, GL_ONE
  float w0 = 1-output[(y*viewport[2]+x)*4+3]; //, w1 = 1; make the compiler happy :)
  output[(y*viewport[2]+x)*4+0] += w0* dst.x;
  output[(y*viewport[2]+x)*4+1] += w0* dst.y;
  output[(y*viewport[2]+x)*4+2] += w0* dst.z;
  output[(y*viewport[2]+x)*4+3] += w0* dst.w;
#endif
}
#endif

template <int ANGLE, int PSI, int SHADING>
static void raycasting_cpu(
        unsigned char *output_rgba8,
        int *viewport, 
        float *invmvp,
        int nPhi, 
        int iPhi,
        int nNodes, 
        int nTriangles,
        float *data, 
        float2 *grad,
        BVHNodeD *bvh,
        float *disp,
        float *invdet,
        int *neighbors,
        float *psi,
        float2 psi_range,
        float2 angle_range,
        float slice_highlight_ratio,
        float3 Ka, 
        float3 Kd, 
        float3 Ks,
        float3 L,
        float4 *tf,
        float4 *ptf,
        float2 trans, 
        float stepsize)
{
  fprintf(stderr, "[volren] CPU rendering, width=%d, height=%d\n", viewport[2], viewport[3]);
#pragma omp parallel for collapse(2)
  for (uint x = 0; x < viewport[2]; x ++) {
    for (uint y = 0; y < viewport[3]; y ++) {
      float3 rayO, rayD;
      setup_ray(viewport, invmvp, x, y, rayO, rayD);

      float4 dst = make_float4(0.f); 
      raycasting<ANGLE, PSI, SHADING>(dst, nPhi, iPhi, nNodes, nTriangles, data, grad, bvh, disp, invdet, neighbors, psi, psi_range, angle_range, 
          slice_highlight_ratio, Ka, Kd, Ks, L, tf, ptf, trans, rayO, rayD, stepsize);

      output_rgba8[(y*viewport[2]+x)*4+0] = clamp(dst.x, 0.f, 1.f) * 255;
      output_rgba8[(y*viewport[2]+x)*4+1] = clamp(dst.y, 0.f, 1.f) * 255;
      output_rgba8[(y*viewport[2]+x)*4+2] = clamp(dst.z, 0.f, 1.f) * 255;
      output_rgba8[(y*viewport[2]+x)*4+3] = clamp(dst.w, 0.f, 1.f) * 255;
    }
  }
}

/////////////////////////////
extern "C" {

void rc_test_point_locator(ctx_rc *ctx, float x, float y)
{
#if WITH_CUDA
  test_point_locator_kernel<<<1, 1>>>(
      (int*)ctx->d_output_rgba8, 
      x, y, ctx->d_bvh, ctx->d_invdet);

  int nid;
  hipMemcpy(&nid, ctx->d_output_rgba8, sizeof(int), hipMemcpyDeviceToHost);
  fprintf(stderr, "[rc_test_point_locator] x={%f, %f}, nid=%d\n", x, y, nid);
#endif
}

void rc_render(ctx_rc *ctx)
{
#if WITH_CUDA
  checkLastCudaError("[rc_render][0]");
  const dim3 blockSize(16, 16); 
  const dim3 gridSize = dim3(iDivUp(ctx->viewport[2], blockSize.x), iDivUp(ctx->viewport[3], blockSize.y));

  hipMemcpy(ctx->d_viewport, ctx->viewport, sizeof(int)*4, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_viewport), ctx->viewport, sizeof(int)*4);
  hipMemcpy(ctx->d_invmvp, ctx->invmvp, sizeof(float)*16, hipMemcpyHostToDevice);
  // hipMemcpyToSymbol(HIP_SYMBOL(c_invmvp), ctx->invmvp, sizeof(float)*16);
 
  raycasting_kernel<1,1,1><<<gridSize, blockSize>>>(
          ctx->d_output_rgba8,
          ctx->d_viewport, 
          ctx->d_invmvp,
          ctx->nPhi, 
          ctx->iPhi, 
          ctx->nNodes,
          ctx->nTriangles,
          ctx->d_data, 
          (float2*)ctx->d_grad,
          ctx->d_bvh,
          ctx->d_disp,
          ctx->d_invdet,
          ctx->d_neighbors, 
          ctx->d_psi,
          make_float2(ctx->psi_range_min, ctx->psi_range_max),
          make_float2(ctx->angle_range_min, ctx->angle_range_max),
          ctx->slice_highlight_ratio,
          make_float3(ctx->Ka), 
          make_float3(ctx->Kd),
          make_float3(ctx->Ks),
          make_float3(ctx->light_direction[0], ctx->light_direction[1], ctx->light_direction[2]),
          (float4*)ctx->d_tf,
          (float4*)ctx->d_ptf,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);

  hipDeviceSynchronize();
  checkLastCudaError("[rc_render]");
#else
  rc_render_cpu(ctx);
#endif
}

void rc_render_cpu(ctx_rc *ctx)
{
  raycasting_cpu<1,1,1>(
          (unsigned char*)ctx->h_output,
          ctx->viewport, 
          ctx->invmvp,
          ctx->nPhi, 
          ctx->iPhi, 
          ctx->nNodes,
          ctx->nTriangles,
          ctx->h_data, 
          (float2*)ctx->h_grad,
          ctx->h_bvh,
          ctx->h_disp,
          ctx->h_invdet,
          ctx->h_neighbors,
          ctx->h_psi,
          make_float2(ctx->psi_range_min, ctx->psi_range_max),
          make_float2(ctx->angle_range_min, ctx->angle_range_max),
          ctx->slice_highlight_ratio,
          make_float3(ctx->Ka), 
          make_float3(ctx->Kd),
          make_float3(ctx->Ks),
          make_float3(ctx->light_direction[0], ctx->light_direction[1], ctx->light_direction[2]),
          (float4*)ctx->h_tf,
          (float4*)ctx->h_ptf,
          make_float2(ctx->trans[0], ctx->trans[1]), 
          ctx->stepsize);
}

void rc_bind_bvh(ctx_rc *ctx, int nBVHNodes, BVHNodeD *bvh)
{
  ctx->nBVHNodes = nBVHNodes;
  ctx->h_bvh = bvh;
#if WITH_CUDA
  if (ctx->d_bvh != NULL)
    hipFree(ctx->d_bvh);

  hipMalloc((void**)&ctx->d_bvh, sizeof(BVHNodeD)*nBVHNodes);
  hipMemcpy(ctx->d_bvh, bvh, sizeof(BVHNodeD)*nBVHNodes, hipMemcpyHostToDevice);
#endif
}

void rc_bind_neighbors(ctx_rc *ctx, int nTriangles, int *neighbors)
{
  int *triangleBVHNodeMap = (int*)malloc(sizeof(int)*nTriangles);
  for (int i=0; i<ctx->nBVHNodes; i++) {
    if (ctx->h_bvh[i].triangleId != -1) 
      triangleBVHNodeMap[ctx->h_bvh[i].triangleId] = i;
  }

  ctx->h_neighbors = (int*)realloc(ctx->h_neighbors, sizeof(int)*nTriangles*3);
  for (int i=0; i<nTriangles; i++) {
    for (int j=0; j<3; j++) {
      if (neighbors[i*3+j] < 0) ctx->h_neighbors[i*3+j] = -1;
      else ctx->h_neighbors[i*3+j] = triangleBVHNodeMap[neighbors[i*3+j]];
    }
  }

  free(triangleBVHNodeMap);

#if WITH_CUDA
  if (ctx->d_neighbors != NULL)
    hipFree(ctx->d_neighbors);

  hipMalloc((void**)&ctx->d_neighbors, sizeof(int)*nTriangles*3);
  hipMemcpy(ctx->d_neighbors, ctx->h_neighbors, sizeof(int)*nTriangles*3, hipMemcpyHostToDevice);
#endif
}

void rc_set_default_tf(ctx_rc *ctx)
{
  float r[3] = {0.7215686274509804, 0.1803921568627451, 0.1803921568627451}, 
        b[3] = {0.2, 0.4, 0.8};

  float *tf = ctx->h_tf;
  for (int i=0; i<size_tf; i++) {
    float x = (float)i / (size_tf-1);
    tf[i*4] = x*b[0] + (1-x)*r[0];
    tf[i*4+1] = x*b[1] + (1-x)*r[1];
    tf[i*4+2] = x*b[2] + (1-x)*r[2];
    tf[i*4+3] = fminf(0.999f, (x-0.5)*(x-0.5)*40);
  }
  rc_set_tf(ctx, tf);
}

void rc_set_tf(ctx_rc *ctx, float *tf) 
{
  if (tf != ctx->h_tf)
    memcpy(ctx->h_tf, tf, sizeof(float)*size_tf*4);
 
#if PREINT
  createPreIntegrationTable(ctx->h_ptf, ctx->h_tf, size_tf);
#endif

#if WITH_CUDA
  hipMemcpy(ctx->d_tf, tf, sizeof(float)*size_tf*4, hipMemcpyHostToDevice);
#if PREINT
  hipMemcpy(ctx->d_ptf, ctx->h_tf, sizeof(float)*size_tf*size_tf*4, hipMemcpyHostToDevice);
#endif
  checkLastCudaError("[rc_set_tf]");
#endif
}

void rc_bind_psi(ctx_rc *ctx, int nNodes, float *psi, float psi_min, float psi_max)
{
  ctx->h_psi = psi;
  ctx->psi_min = psi_min;
  ctx->psi_max = psi_max;
#if WITH_CUDA
  if (ctx->d_psi == NULL)
    hipMalloc((void**)&ctx->d_psi, sizeof(float)*nNodes);
  hipMemcpy(ctx->d_psi, psi, sizeof(float)*nNodes, hipMemcpyHostToDevice);
#endif
}

void rc_set_psi_range(ctx_rc *ctx, bool on, float psi_range_min, float psi_range_max)
{
  ctx->toggle_psi_range = on;
  if (on) {
    ctx->psi_range_min = psi_range_min; 
    ctx->psi_range_max = psi_range_max;
  } else {
    ctx->psi_range_min = ctx->psi_min; 
    ctx->psi_range_max = ctx->psi_max;
  }
}

void rc_set_shading(ctx_rc *ctx, bool on, float Ka, float Kd, float Ks, float lx, float ly, float lz)
{
  // fprintf(stderr, "shading: Ka=%f, Kd=%f, Ks=%f, L={%f, %f, %f}\n", 
  //     Ka, Kd, Ks, lx, ly, lz);

  if (!on) {Ka = Kd = Ks = 0;}

  ctx->toggle_shading = on;
  ctx->Ka = Ka;
  ctx->Kd = Kd;
  ctx->Ks = Ks;
  ctx->light_direction[0] = lx;
  ctx->light_direction[1] = ly;
  ctx->light_direction[2] = lz;
}

void rc_set_angle_range(ctx_rc *ctx, bool on, float angle_range_min, float angle_range_max)
{
  // fprintf(stderr, "angle_range: %d, %f, %f\n", on, angle_range_min, angle_range_max);
  ctx->toggle_angle_range = on;
  if (on) {
    ctx->angle_range_min = angle_range_min;
    ctx->angle_range_max = angle_range_max;
  } else {
    ctx->angle_range_min = -10;
    ctx->angle_range_max = 10;
  }
}

void rc_set_slice_highlight_ratio(ctx_rc *ctx, bool on, float ratio)
{
  ctx->toggle_slice_highlight = on;
  if (on) 
    ctx->slice_highlight_ratio = ratio;
  else 
    ctx->slice_highlight_ratio = 1;
}

void rc_bind_disp(ctx_rc *ctx, int nNodes, float *disp)
{
  ctx->h_disp = disp;
#if WITH_CUDA
  if (ctx->d_disp == NULL)
    hipMalloc((void**)&ctx->d_disp, sizeof(float)*nNodes*2);
  hipMemcpy(ctx->d_disp, disp, sizeof(float)*nNodes*2, hipMemcpyHostToDevice);
#endif
}

void rc_bind_invdet(ctx_rc *ctx, int nTriangles, float *invdet)
{
  ctx->h_invdet = invdet;
#if WITH_CUDA
  if (ctx->d_invdet == NULL)
    hipMalloc((void**)&ctx->d_invdet, sizeof(float)*nTriangles);
  hipMemcpy(ctx->d_invdet, invdet, sizeof(float)*nTriangles, hipMemcpyHostToDevice);
  
  checkLastCudaError("[rc_bind_invdet]");
#endif
}

void rc_bind_data(ctx_rc *ctx, int nNodes, int nTriangles, int nPhi, int iPhi, float *data, float *grad)
{
  ctx->h_data = data;
  ctx->h_grad = grad;
  ctx->nNodes = nNodes;
  ctx->nPhi = nPhi;
  ctx->iPhi = iPhi;
  ctx->nTriangles = nTriangles;
#if WITH_CUDA
  if (ctx->d_data == NULL)
    hipMalloc((void**)&ctx->d_data, sizeof(float)*nNodes*nPhi);
  hipMemcpy(ctx->d_data, data, sizeof(float)*nNodes*nPhi, hipMemcpyHostToDevice);
  if (ctx->d_grad == NULL)
    hipMalloc((void**)&ctx->d_grad, sizeof(float)*nPhi*nTriangles*2);
  hipMemcpy(ctx->d_grad, grad, sizeof(float)*nPhi*nTriangles*2, hipMemcpyHostToDevice);
  
  checkLastCudaError("[rc_bind_data]");
#endif
}

void rc_create_ctx(ctx_rc **ctx)
{
  *ctx = (ctx_rc*)malloc(sizeof(ctx_rc));
  memset(*ctx, 0, sizeof(ctx_rc));

  (*ctx)->h_tf = (float*)malloc(sizeof(float)*size_tf*4);
  (*ctx)->h_ptf = (float*)malloc(sizeof(float)*size_tf*size_tf*4);

  const size_t max_npx = 4096*4096;
  (*ctx)->h_output = malloc(4*max_npx);

  (*ctx)->slice_highlight_ratio = 1.0;

#if WITH_CUDA
  hipSetDevice(0);
  hipMalloc((void**)&((*ctx)->d_output_rgba8), 4*max_npx); 
  
  hipMalloc((void**)&((*ctx)->d_tf), sizeof(float)*size_tf*4);
  hipMalloc((void**)&((*ctx)->d_ptf), sizeof(float)*size_tf*size_tf*4);
  hipMalloc((void**)&((*ctx)->d_viewport), sizeof(int)*4);
  hipMalloc((void**)&((*ctx)->d_invmvp), sizeof(float)*16);
  
  checkLastCudaError("[rc_init]");
#endif
}

void rc_destroy_ctx(ctx_rc **ctx)
{
  // TODO: free any resources
#if WITH_CUDA
  hipFree((*ctx)->d_output_rgba8);
  hipFree((*ctx)->d_tf);
  hipFree((*ctx)->d_ptf);
  hipFree((*ctx)->d_disp);
  hipFree((*ctx)->d_invdet);
  hipFree((*ctx)->d_psi);
  hipFree((*ctx)->d_neighbors);
#endif
  free((*ctx)->h_neighbors);
  free((*ctx)->h_output);
  free((*ctx)->h_tf);
  free((*ctx)->h_ptf);
  free(*ctx); 
  *ctx = NULL; 
}

void rc_set_viewport(ctx_rc *ctx, int x, int y, int w, int h)
{
  ctx->viewport[0] = x; 
  ctx->viewport[1] = y; 
  ctx->viewport[2] = w; 
  ctx->viewport[3] = h; 
}

void rc_set_range(ctx_rc *ctx, float a, float b)
{
  float c = 1.f/(b-a);
  ctx->trans[0] = c; 
  ctx->trans[1] = -a*c; 
}

void rc_set_stepsize(ctx_rc *ctx, float stepsize)
{
  ctx->stepsize = stepsize;
}

void rc_set_invmvpf(ctx_rc *ctx, float *invmvp)
{
  memcpy(ctx->invmvp, invmvp, sizeof(float)*16); 
}

void rc_set_invmvpd(ctx_rc *ctx, double *invmvp)
{
  for (int i=0; i<16; i++) {
    ctx->invmvp[i] = invmvp[i]; 
  }
}

void rc_clear_output(ctx_rc *ctx)
{
  memset(ctx->h_output, 0, ctx->viewport[2]*ctx->viewport[3]*4);
#if WITH_CUDA
  hipMemset(ctx->d_output_rgba8, 0, 4*ctx->viewport[2]*ctx->viewport[3]);
#endif
}

void rc_copy_output_to_host(ctx_rc *ctx)
{
#if WITH_CUDA
  hipMemcpy(ctx->h_output, ctx->d_output_rgba8, 4*ctx->viewport[2]*ctx->viewport[3], hipMemcpyDeviceToHost);
#if 0
  unsigned char* img = (unsigned char*)ctx->h_output;
  const int j = 0;
  for (int i=0; i<720; i++) {
    fprintf(stderr, "%f\n", img[(j*720+i)*4+3]);
  }
#endif
#endif
}

} // extern "C" 
/////////////////

#include "hip/hip_runtime.h"
#include "volren.cuh"

bool QuadNodeD_insideQuad(const QuadNodeD &q, float x, float y)
{
  return x >= q.Ax && x < q.Bx && y >= q.Ay && y < q.By;
}

bool QuadNodeD_insideTriangle(const QuadNodeD &q, float x, float y, float &alpha, float &beta, float &gamma) 
{
  alpha = ((q.y1 - q.y2)*(x - q.x2) + (q.x2 - q.x1)*(y - q.y2)) /
          ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  beta = ((q.y2 - q.y0)*(x - q.x2) + (q.x0 - q.x2)*(y - q.y2)) /
         ((q.y1 - q.y2)*(q.x0 - q.x2) + (q.x2 - q.x1)*(q.y0 - q.y2));
  gamma = 1.0 - alpha - beta;
  // fprintf(stderr, "barycentric=%f, %f, %f\n", alpha, beta, gamma);
  return alpha >= 0 && beta >= 0 && gamma >= 0;
}

int QuadNodeD_locatePoint_recursive(const QuadNodeD *q, const QuadNodeD *nodes, float x, float y, float &alpha, float &beta, float &gamma)
{
  if (q->triangleId >= 0) { //leaf node
    bool succ = QuadNodeD_insideTriangle(*q, x, y, alpha, beta, gamma);
    if (succ) return q->triangleId;
  } else if (QuadNodeD_insideQuad(*q, x, y)) {
    for (int j=0; j<4; j++) {
      if (q->childrenIds[j] > 0) {
        int result = QuadNodeD_locatePoint_recursive(&nodes[q->childrenIds[j]], nodes, x, y, alpha, beta, gamma);
        if (result >= 0) return result;
      }
    }
  }
  return -1;
}

int QuadNodeD_locatePoint(QuadNodeD *nodes, float x, float y, float &alpha, float &beta, float &gamma)
{
  // float alpha, beta, gamma;
  static const int maxStackSize = 64;
  int stack[maxStackSize];
  int stackPos = 0;
  stack[stackPos++] = 0; // push root

  while (stackPos > 0) {
    const int i = stack[--stackPos]; // pop
    const QuadNodeD &q = nodes[i];

    // fprintf(stderr, "D_checking node %d, %f, %f, %f, %f\n", i, q.Ax, q.Ay, q.Bx, q.By);
    // fprintf(stderr, "D_checking node %d\n", i);

    if (q.triangleId >= 0) { // leaf node
      bool succ = QuadNodeD_insideTriangle(q, x, y, alpha, beta, gamma);
      if (succ) return q.triangleId;
    } else if (QuadNodeD_insideQuad(q, x, y)) { // non-leaf node
      for (int j=0; j<4; j++) {
        if (q.childrenIds[j] > 0)
          stack[stackPos++] = q.childrenIds[j];
      }
    }
  }
  return -1;
}

float QuadNodeD_sample(QuadNodeD *nodes, float x, float y, float *scalar) {
  float alpha, beta, gamma;
  int i = QuadNodeD_locatePoint(nodes, x, y, alpha, beta, gamma);
  const QuadNodeD &q = nodes[i];

  return alpha * scalar[q.i0] 
    + beta * scalar[q.i1]
    + gamma * scalar[q.i2];
}
